#include "hip/hip_runtime.h"
/***********************************************************************[pfcualloc.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "pfsolve.h"
#include "pfdevice.cuh"
#include <cub/device/device_scan.cuh>
#include <cub/device/device_select.cuh>
using namespace hipcub;

namespace pFROST {

	namespace SIGmA {
		//=============================//
		//	  CUDA memory management   //
		//=============================//
		const size_t hc_srsize = sizeof(S_REF);
		const size_t hc_scsize = sizeof(SCLAUSE);
		const size_t hc_otsize = sizeof(OT);
		const size_t hc_olsize = sizeof(OL);
		const size_t hc_cnfsize = sizeof(CNF);
		const size_t hc_varsize = sizeof(uint32);
		const size_t hc_gstsize = sizeof(GSTATS);
		const size_t hc_cuvecsize = sizeof(cuVecU);

		__global__ void resizeCNF_k(CNF* cnf, CNF* hcnf) { cnf->resize(hcnf->data().size, hcnf->size()); }
		__global__ void resizeCNF_k(CNF* cnf, S_REF d_size, uint32 cs_size) { cnf->resize(d_size, cs_size); }
		__global__ void scatter_k(CNF* src, S_REF* scatter, addr_t stencil) {
			uint32 tid = global_tx();
			while (tid < src->size()) {
				SCLAUSE& c = src->clause(tid);
				if (c.deleted()) stencil[tid] = 0, scatter[tid] = 0;
				else stencil[tid] = 1, scatter[tid] = (c.size() - 1) + dc_nbuckets;
				tid += stride_x();
			}
		}
		__global__ void compact_k(CNF* src, CNF* dest, S_REF* scatter, addr_t stencil) {
			uint32 tid = global_tx();
			while (tid < src->size()) {
				if (stencil[tid]) {
					S_REF new_r = scatter[tid];
					new (dest->cref(new_r)) SCLAUSE(src->clause(tid));
					assert(src->clause(tid).size() == dest->cref(new_r)->size());
				}
				tid += stride_x();
			}
		}
		__global__ void assignListPtrs(OT* ot, uint32* hist, S_REF* segs, uint32 size)
		{
			uint32 tid = global_tx();
			while (tid < size) {
				assert(segs[tid] < UINT32_MAX);
				(*ot)[tid].alloc(ot->data(segs[tid]), hist[tid]);
				tid += stride_x();
			}
		}
		
		void cuMM::compactCNF(CNF* src, CNF* dest) {
			uint32 old_size = pinned_cnf->size();
			assert(old_size <= nscatters);
			assert(hc_nbuckets == sizeof(SCLAUSE) / sizeof(uint32));
			PFLOGN2(2, " Compacting simplified CNF (%d to %d) on GPU..", old_size, inf.nClauses);
			S_REF data_size = inf.nClauses * hc_nbuckets + (inf.nLiterals - inf.nClauses);
			resizeCNF_k << <1, 1 >> > (dest, data_size, inf.nClauses);
			size_t tb = 0, ftb = 0;
			uint32* ts = d_lits;
			uint32 nTereads = BLOCK1D, maxBlocks = maxGPUTereads / nTereads;
			uint32 nBlocks = std::min((old_size + nTereads - 1) / nTereads, maxBlocks);
			scatter_k << <nBlocks, nTereads >> > (src, d_scatter, d_stencil);
			DeviceScan::ExclusiveSum(NULL, tb, d_scatter, d_scatter, old_size), assert(tb <= litsbytes);
			DeviceScan::ExclusiveSum(ts, tb, d_scatter, d_scatter, old_size);
			DeviceSelect::Flagged(NULL, ftb, d_scatter, d_stencil, d_cs_mem, ts, old_size), assert(ftb <= litsbytes);
			DeviceSelect::Flagged(ts + 1, ftb, d_scatter, d_stencil, d_cs_mem, ts, old_size);
			compact_k << <nBlocks, nTereads >> > (src, dest, d_scatter, d_stencil);
			pinned_cnf->resize(data_size, inf.nClauses);
			LOGERR(" CNF compact failed");
			sync();
			PFLDONE(2, 5);
		}

		bool cuMM::allocVars(VARS*& vars, const size_t& resCap) {
			assert(vars == NULL);
			assert(resCap && resCap <= UINT32_MAX);
			vars = new VARS();
			size_t uintVec_sz = inf.maxVar * hc_varsize;
			size_t varsize = inf.maxVar + 1;
			size_t scores_sz = varsize * hc_varsize;
			size_t resolved_sz = resCap * hc_varsize;
			size_t newCap = hc_gstsize + scores_sz + resolved_sz + (hc_cuvecsize + uintVec_sz) * 3;
			varsPool.cap = newCap, cap += newCap;
			assert(varsPool.cap);
			if (!hasUnifiedMem("Fixed")) { cap -= newCap; return false; }
			CHECK(hipMallocManaged((void**)&varsPool.mem, varsPool.cap));
			addr_t ea = varsPool.mem, end = ea + varsPool.cap;
			vars->gstats = (GSTATS*)ea, ea += hc_gstsize;
			vars->pVars = (cuVecU*)ea, ea += hc_cuvecsize;
			vars->units = (cuVecU*)ea, ea += hc_cuvecsize;
			vars->resolved = (cuVecU*)ea, ea += hc_cuvecsize;
			uint32* uintPtr = (uint32*)ea;
			vars->pVars->alloc(uintPtr, inf.maxVar), uintPtr += inf.maxVar; d_units = uintPtr;
			vars->units->alloc(uintPtr, inf.maxVar), uintPtr += inf.maxVar;
			vars->eligible = uintPtr, uintPtr += inf.maxVar;
			vars->scores = uintPtr, uintPtr += varsize;
			vars->resolved->alloc(uintPtr, uint32(resCap)), uintPtr += resCap;
			assert((addr_t)uintPtr == end);
			if (devProp.major > 5) {
				PFLOGN2(2, " Advising GPU driver to favor global over system memory..");
				CHECK(hipMemAdvise(vars->gstats, hc_gstsize, hipMemAdviseSetPreferredLocation, MASTER_GPU));
				addr_t tmpPtr = ea + uintVec_sz; // skip pVars
				CHECK(hipMemAdvise(tmpPtr, end - tmpPtr, hipMemAdviseSetPreferredLocation, MASTER_GPU));
				CHECK(hipMemPrefetchAsync(tmpPtr, end - tmpPtr, MASTER_GPU));
				PFLDONE(2, 5);
			}
			if (pinned_units == NULL) {
				CHECK(hipHostAlloc((void**)&pinned_units, uintVec_sz, hipHostMallocDefault));
				vars->cachedUnits = pinned_units;
			}
			else vars->cachedUnits = pinned_units;
			return true;
		}

		bool cuMM::allocHist(cuHist& cuhist, const size_t& litsCap)
		{
			assert(litsCap && litsCap <= UINT32_MAX);
			assert(inf.nDualVars == V2L(inf.maxVar + 1ULL));
			litsbytes = litsCap * hc_varsize;
			size_t segBytes = inf.nDualVars * hc_srsize;
			size_t histBytes = inf.nDualVars * hc_varsize;
			size_t newCap = segBytes + histBytes + litsbytes;
			assert(newCap);
			if (hhistPool.cap < histBytes) {
				if (hhistPool.cap) {
					assert(hhistPool.mem != NULL);
					assert(cuhist.h_hist != NULL);
					CHECK(hipHostFree(hhistPool.mem));
					hhistPool.mem = NULL;
				}
				assert(hhistPool.mem == NULL);
				CHECK(hipHostAlloc((void**)&hhistPool.mem, histBytes, hipHostMallocDefault));
				cuhist.h_hist = (uint32*)hhistPool.mem;
				hhistPool.cap = histBytes;
			}
			if (histPool.cap < newCap) {
				if (histPool.cap) {
					assert(histPool.mem != NULL);
					assert(d_hist != NULL);
					assert(d_segs != NULL);
					assert(cuhist.d_hist != NULL);
					assert(cuhist.d_segs != NULL);
					assert(cuhist.d_lits != NULL);
					CHECK(hipFree(histPool.mem));
					histPool.mem = NULL;
					dcap -= histPool.cap;
					assert(dcap >= 0);
					_free += histPool.cap;
				}
				assert(histPool.mem == NULL);
				dcap += newCap;
				if (!hasDeviceMem("Histogram")) { dcap -= newCap; return false; }
				CHECK(hipMalloc((void**)&histPool.mem, newCap));
				// NOTE: d_segs, d_hist used internally by OT allocation and externally by BVE for calculating resolvents offsets (memory reuse)
				//		 d_lits is used as temporary storage as well for CUB routines
				addr_t ea = histPool.mem;
				cuhist.d_segs = d_segs = (S_REF*)ea, ea += segBytes;
				cuhist.d_hist = d_hist = (uint32*)ea, ea += histBytes;
				cuhist.d_lits = d_lits = (uint32*)ea, ea += litsbytes;
				assert(ea == histPool.mem + newCap);
				cuhist.thrust_hist = t_iptr(d_hist);
				cuhist.thrust_lits = t_iptr(d_lits);
				histPool.cap = newCap;
			}
			return true;
		}

		bool cuMM::allocAux(const size_t& clsCap)
		{
			assert(clsCap && clsCap <= UINT32_MAX);
			nscatters = clsCap;
			size_t scatterBytes = nscatters * hc_srsize;
			size_t newCap = scatterBytes + nscatters;
			assert(newCap);
			if (pinned_cnf == NULL) 
				CHECK(hipHostAlloc((void**)&pinned_cnf, hc_cnfsize, hipHostMallocDefault));
			if (auxPool.cap < newCap) {
				if (auxPool.cap) {
					assert(auxPool.mem != NULL);
					assert(d_scatter != NULL);
					CHECK(hipFree(auxPool.mem));
					auxPool.mem = NULL;
					dcap -= auxPool.cap;
					assert(dcap >= 0);
					_free += auxPool.cap;
				}
				assert(auxPool.mem == NULL);
				dcap += newCap;
				if (!hasDeviceMem("Auxiliary ")) { dcap -= newCap; return false; }
				CHECK(hipMalloc((void**)&auxPool.mem, newCap));
				d_scatter = (S_REF*)auxPool.mem;
				d_stencil = auxPool.mem + scatterBytes;
				auxPool.cap = newCap;
			}
			return true;
		}

		bool cuMM::resizeCNF(CNF*& cnf, const size_t& clsCap, const size_t& litsCap) {
			assert(clsCap && clsCap <= UINT32_MAX);
			assert(litsCap && litsCap <= UINT32_MAX);
			assert(litsCap >= clsCap);
			size_t csBytes = clsCap * hc_srsize;
			size_t dataBytes = clsCap * hc_scsize + (litsCap - clsCap) * hc_bucket;
			assert(dataBytes % hc_bucket == 0);
			size_t newCap = hc_cnfsize + dataBytes + csBytes;
			assert(newCap);
			if (cnfPool.cap == 0) {
				assert(cnf == NULL);
				assert(cnfPool.mem == NULL);
				cnfPool.cap = newCap, cap += cnfPool.cap;
				if (!hasUnifiedMem("CNF")) { cap -= newCap; return false; }
				CHECK(hipMallocManaged((void**)&cnfPool.mem, cnfPool.cap));
				if (devProp.major > 5) {
					PFLOGN2(2, " Advising GPU driver to favor global over system memory..");
					CHECK(hipMemAdvise(cnfPool.mem, cnfPool.cap, hipMemAdviseSetPreferredLocation, MASTER_GPU));
					PFLDONE(2, 5);
				}
				cnf = (CNF*)cnfPool.mem;
				S_REF data_cap = S_REF(dataBytes / hc_bucket);
				new (cnf) CNF(data_cap, uint32(clsCap));
				d_cnf_mem = cnf->data().mem, d_cs_mem = cnf->csData();
			}
			else {
				assert(cnf != NULL);
				assert(cnfPool.mem != NULL);
				cap -= cnfPool.cap;
				assert(cap >= 0);
				_free += cnfPool.cap;
				cap += newCap;
				if (!hasUnifiedMem("CNF")) { cap -= newCap; return false; }
				cacheCNFPtr(cnf);
				addr_t newMem = NULL;
				CHECK(hipMallocManaged((void**)&newMem, newCap));
				sync();
				if (devProp.major > 5) {
					PFLOGN2(2, " Advising GPU driver to favor global over system memory..");
					CHECK(hipMemAdvise(newMem, newCap, hipMemAdviseSetPreferredLocation, MASTER_GPU));
					CHECK(hipMemPrefetchAsync(newMem, newCap, MASTER_GPU));
					PFLDONE(2, 5);
				}
				CNF* tmp_cnf = (CNF*)newMem;
				S_REF data_cap = S_REF(dataBytes / hc_bucket);
				new (tmp_cnf) CNF(data_cap, uint32(clsCap));
				d_cnf_mem = tmp_cnf->data().mem, d_cs_mem = tmp_cnf->csData();
				if (profile_gpu) cutimer->start();
				if (gc_par) compactCNF(cnf, tmp_cnf);
				else {
					sync(), tmp_cnf->copyFrom(cnf);
					pinned_cnf->resize(tmp_cnf->data().size, tmp_cnf->size());
				}
				if (profile_gpu) cutimer->stop(), cutimer->gc += cutimer->gpuTime();
				CHECK(hipFree(cnfPool.mem));
				cnfPool.mem = newMem, cnfPool.cap = newCap, cnf = tmp_cnf;
			}
			return true;
		}

		bool cuMM::resizeOTAsync(OT*& ot, const size_t& litsCap, const hipStream_t& _s) {
			assert(d_hist != NULL);
			assert(d_segs != NULL);
			assert(litsCap && litsCap <= UINT32_MAX);
			size_t tb = 0;
			DeviceScan::ExclusiveSum(NULL, tb, d_hist, d_segs, inf.nDualVars, _s), assert(tb <= litsbytes);
			DeviceScan::ExclusiveSum(d_lits, tb, d_hist, d_segs, inf.nDualVars, _s);
			if (!otBlocks) otBlocks = std::min((inf.nDualVars + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			size_t newCap = hc_otsize + inf.nDualVars * hc_olsize + litsCap * hc_srsize;
			assert(newCap);
			if (otPool.cap < newCap) { // realloc
				if (otPool.cap) {
					assert(otPool.mem != NULL);
					assert(ot != NULL);
					CHECK(hipFree(otPool.mem));
					otPool.mem = NULL;
					cap -= otPool.cap;
					assert(cap >= 0);
					_free += otPool.cap;
				}
				assert(otPool.mem == NULL);
				cap += newCap;
				if (!hasUnifiedMem("OT")) { cap -= newCap; return false; }
				CHECK(hipMallocManaged((void**)&otPool.mem, newCap));
				if (devProp.major > 5) {
					PFLOGN2(2, " Advising GPU driver to favor global over system memory..");
					CHECK(hipMemAdvise(otPool.mem, newCap, hipMemAdviseSetPreferredLocation, MASTER_GPU));
					CHECK(hipMemPrefetchAsync(otPool.mem, newCap, MASTER_GPU, _s));
					PFLDONE(2, 5);
				}
				ot = (OT*)otPool.mem;
				LOGERR("Summing histogram failed");
				sync(_s); // needed for calling the next constructor on host
				new (ot) OT(inf.nDualVars);
				assignListPtrs << <otBlocks, BLOCK1D, 0, _s >> > (ot, d_hist, d_segs, inf.nDualVars);
				otPool.cap = newCap;
			}
			else 
				assignListPtrs << <otBlocks, BLOCK1D, 0, _s >> > (ot, d_hist, d_segs, inf.nDualVars);
			if (sync_always) {
				LOGERR("Occurrence lists allocation failed");
				sync(_s);
			}
			return true;
		}

		void cuMM::createMirror(CNF*& hcnf, const size_t& clsCap, const size_t& litsCap)
		{
			assert(clsCap && clsCap <= UINT32_MAX);
			assert(litsCap && litsCap <= UINT32_MAX);
			assert(litsCap >= clsCap);
			size_t csBytes = clsCap * hc_srsize;
			size_t dataBytes = clsCap * hc_scsize + (litsCap - clsCap) * hc_bucket;
			assert(dataBytes % hc_bucket == 0);
			size_t newCap = hc_cnfsize + dataBytes + csBytes;
			assert(newCap);
			if (hcnfPool.cap < newCap) {
				hcnfPool.cap = newCap;
				pfalloc(hcnfPool.mem, hcnfPool.cap);
				hcnf = (CNF*)hcnfPool.mem;
			}
			S_REF data_cap = S_REF(dataBytes / hc_bucket);
			new (hcnf) CNF(data_cap, uint32(clsCap));
		}

		void cuMM::mirrorCNF(CNF*& hcnf)
		{
			assert(cnfPool.cap);
			assert(cnfPool.mem != NULL);
			CHECK(hipMemcpy(hcnf, cnfPool.mem, hc_cnfsize, hipMemcpyDeviceToHost));
			size_t csBytes = hcnf->size() * hc_srsize;
			size_t dataBytes = hcnf->data().size * hc_bucket;
			size_t newCap = hc_cnfsize + dataBytes + csBytes;
			assert(newCap <= cnfPool.cap);
			if (hcnfPool.cap < newCap) {
				hcnfPool.cap = newCap;
				pfalloc(hcnfPool.mem, hcnfPool.cap);
				hcnf = (CNF*)hcnfPool.mem;
			}
			hcnf->fixPointer(); // replace device with host pointers
		}

		void cuMM::resizeCNFAsync(CNF* dcnf, const S_REF& data_size, const uint32& cs_size)
		{
			assert(dcnf != NULL);
			assert(data_size);
			assert(cs_size);
			resizeCNF_k << <1, 1 >> > (dcnf, data_size, cs_size);
			if (sync_always) {
				LOGERR("Resizing CNF failed");
				sync();
			}
		}

		void cuMM::freeVars() {
			if (varsPool.mem != NULL) {
				d_units = NULL;
				CHECK(hipFree(varsPool.mem)), varsPool.mem = NULL;
				cap -= varsPool.cap;
				assert(cap >= 0);
				_free += varsPool.cap;
				varsPool.cap = 0;
			}
		}

		void cuMM::freeCNF() {
			if (cnfPool.mem != NULL) {
				d_cnf_mem = NULL, d_cs_mem = NULL;
				CHECK(hipFree(cnfPool.mem)), cnfPool.mem = NULL;
				cap -= cnfPool.cap;
				assert(cap >= 0);
				_free += cnfPool.cap;
				cnfPool.cap = 0;
			}
		}

		void cuMM::freeOT() {
			if (otPool.mem != NULL) {
				CHECK(hipFree(otPool.mem)), otPool.mem = NULL;
				cap -= otPool.cap;
				assert(cap >= 0);
				_free += otPool.cap;
				otPool.cap = 0;
			}
		}

		void cuMM::freeFixed() {
			if (auxPool.mem != NULL) {
				CHECK(hipFree(auxPool.mem)), auxPool.mem = NULL;
				_free += auxPool.cap, auxPool.cap = 0;
			}
			if (histPool.mem != NULL) {
				CHECK(hipFree(histPool.mem)), histPool.mem = NULL;
				_free += histPool.cap, histPool.cap = 0;
			}
			dcap = 0;
		}

		void cuMM::freePinned() {
			if (pinned_cnf != NULL) CHECK(hipHostFree(pinned_cnf)), pinned_cnf = NULL;
			if (pinned_units != NULL) CHECK(hipHostFree(pinned_units)), pinned_units = NULL;
			if (hhistPool.mem != NULL) {
				CHECK(hipHostFree(hhistPool.mem)), hhistPool.mem = NULL;
				hhistPool.cap = 0;
			}
		}

		void cuMM::breakMirror() {
			if (hcnfPool.mem != NULL) {
				std::free(hcnfPool.mem), hcnfPool.mem = NULL;
				hcnfPool.cap = 0;
			}
		}

		void TCA::destroy() {
			for (freeBlock_t::iterator i = freeBlocks.begin(); i != freeBlocks.end(); i++) {
				thrust::cuda_cub::free(thrust::cuda_cub::pointer<void>(i->second));
				i->second = NULL;
			}
			for (allocBlock_t::iterator i = allocBlocks.begin(); i != allocBlocks.end(); i++)
				thrust::cuda_cub::free(thrust::cuda_cub::pointer<void>(i->first));
			freeBlocks.clear();
			allocBlocks.clear();
			used = 0;
		}

		char* TCA::allocate(int64 new_cap) {
			char* result = NULL;
			freeBlock_t::iterator freeBlock = freeBlocks.lower_bound(new_cap);
			// found free block
			if (freeBlock != freeBlocks.end()) {
				result = freeBlock->second;
				new_cap = freeBlock->first;
				freeBlocks.erase(freeBlock);
			}
			// no free blocks, allocate new one
			else {
				try { 
					result = thrust::cuda_cub::malloc<char>(new_cap).get(); 
					used += new_cap;
				}
				catch (std::runtime_error&) { 
					PFLOGE("cannot allocate new memory block for Thrust.");
					throw;
				}
			}
			assert(result);
			allocBlocks.insert(std::make_pair(result, new_cap)); // cache new block
			return result;
		}

		void TCA::deallocate(char* ptr, size_t) {
			allocBlock_t::iterator allocBlock = allocBlocks.find(ptr);
			if (allocBlock == allocBlocks.end()) throw INVALID_PTR(ptr);
			int64 new_cap = allocBlock->second;
			allocBlocks.erase(allocBlock);
			freeBlocks.insert(std::make_pair(new_cap, ptr)); // cache free block
		}
	}
}