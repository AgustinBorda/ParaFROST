#include "hip/hip_runtime.h"
/***********************************************************************[pfkernels.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "pfdevice.cuh"
#include "pfbve.cuh"
#include "pfhse.cuh"

namespace pFROST {

	namespace SIGmA {

		template<class T>
		__global__ void memset_k(T* mem, T val, size_t size)
		{
			size_t tid = global_tx();
			while (tid < size) { mem[tid] = val; tid += stride_x(); }
		}

		__global__ void reset_stats(GSTATS* gstats) { gstats->numDelVars = 0, gstats->numClauses = 0, gstats->numLits = 0; }

		__global__ void reset_ot_k(OT* ot)
		{
			uint64 tid = global_tx();
			while (tid < ot->size()) { (*ot)[tid].clear(); tid += stride_x(); }
		}

		__global__ void reduce_ot(CNF* cnf, OT* ot)
		{
			uint64 tid = global_tx();
			while (tid < ot->size()) { reduceOL(*cnf, (*ot)[tid]); tid += stride_x(); }
		}

		__global__ void reduce_ot_p(CNF* cnf, OT* ot, cuVecU* pVars)
		{
			uint32 tid = global_tx();
			while (tid < pVars->size()) {
				assert(pVars->at(tid));
				reduceOL(*cnf, (*ot)[V2D(pVars->at(tid))]);
				tid += stride_x();
			}
		}

		__global__ void reduce_ot_n(CNF* cnf, OT* ot, cuVecU* pVars)
		{
			uint32 tid = global_tx();
			while (tid < pVars->size()) {
				assert(pVars->at(tid));
				reduceOL(*cnf, (*ot)[NEG(V2D(pVars->at(tid)))]);
				tid += stride_x();
			}
		}

		__global__ void create_ot_k(CNF* cnf, OT* ot)
		{
			uint32 tid = global_tx();
			while (tid < cnf->size()) {
				S_REF r = cnf->ref(tid);
				SCLAUSE& c = (*cnf)[r];
				if (c.original() || c.learnt()) {
					uint32* lit = c, * cend = c.end();
#pragma unroll
					while (lit != cend) (*ot)[*lit++].insert(r);
				}
				tid += stride_x();
			}
		}

		__global__ void assign_scores(uint32* eligible, uint32* scores, uint32* hist, uint32 size)
		{
			uint32 tid = global_tx();
			while (tid < size) {
				uint32 v = tid + 1;
				uint32 p = V2D(v), ps = hist[p], ns = hist[NEG(p)];
				eligible[tid] = v;
				scores[v] = rscore(ps, ns);
				tid += stride_x();
			}
		}

		__global__ void assign_scores(uint32* eligible, uint32* scores, uint32* hist, OT* ot, uint32 size)
		{
			uint32 tid = global_tx();
			while (tid < size) {
				uint32 v = tid + 1;
				uint32 p = V2D(v), n = NEG(p), ps = (*ot)[p].size(), ns = (*ot)[n].size();
				hist[p] = ps, hist[n] = ns;
				eligible[tid] = v;
				scores[v] = rscore(ps, ns);
				tid += stride_x();
			}
		}

		__global__ void calc_sig_k(CNF* cnf, uint32 offset, uint32 size)
		{
			uint32 tid = global_tx() + offset;
			while (tid < size) { calcSig(cnf->clause(tid)); tid += stride_x(); }
		}

		__global__ void copy_if_k(uint32* dest, CNF* src, GSTATS* gstats)
		{
			uint32 tid = global_tx();
			while (tid < src->size()) {
				SCLAUSE& c = src->clause(tid);
				if (c.original() || c.learnt()) {
					uint32* d = dest + atomicAdd(&gstats->numLits, c.size());
					uint32* s = c, *cend = c.end();
#pragma unroll
					while (s != cend) *d++ = *s++;
				}
				tid += stride_x();
			}
		}

		__global__ void copy_if_k(CNF* dest, CNF* src)
		{
			uint32 i = global_tx();
			while (i < src->size()) {
				SCLAUSE& s = src->clause(i);
				if (s.original() || s.learnt())
					dest->insert(s);
				i += stride_x();
			}
		}

		__global__ void cnt_reds(CNF* cnf, GSTATS* gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32* sh_rLits = sh_rCls + blockDim.x;
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt())
					nCls++, nLits += c1.size();
				if (tid + blockDim.x < cnf->size()) {
					SCLAUSE& c2 = cnf->clause(tid + blockDim.x);
					if (c2.original() || c2.learnt())
						nCls++, nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rCls, nCls, sh_rLits, nLits);
			warpReduce(sh_rCls, nCls, sh_rLits, nLits);
			if (threadIdx.x == 0) {
				atomicAdd(&gstats->numClauses, nCls);
				atomicAdd(&gstats->numLits, nLits);
			}
		}

		__global__ void cnt_cls(CNF* cnf, GSTATS* gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			while (tid < cnf->size()) {
				SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nCls++;
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nCls++;
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, cnf->size());
			sharedReduce(sh_rCls, nCls);
			warpReduce(sh_rCls, nCls);
			if (threadIdx.x == 0) atomicAdd(&gstats->numClauses, nCls);
		}

		__global__ void cnt_lits(CNF* cnf, GSTATS* gstats)
		{
			uint32* sh_rLits = SharedMemory<uint32>();
			uint32 tid = global_tx_off();
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nLits += c1.size();
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rLits, nLits);
			warpReduce(sh_rLits, nLits);
			if (threadIdx.x == 0) atomicAdd(&gstats->numLits, nLits);
		}

		__global__ void cnt_cls_lits(CNF* cnf, GSTATS* gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32* sh_rLits = sh_rCls + blockDim.x;
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nCls++, nLits += c1.size();
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nCls++, nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rCls, nCls, sh_rLits, nLits);
			warpReduce(sh_rCls, nCls, sh_rLits, nLits);
			if (threadIdx.x == 0) {
				atomicAdd(&gstats->numClauses, nCls);
				atomicAdd(&gstats->numLits, nLits);
			}
		}

		__global__ void ve_k(CNF* cnf, OT* ot, cuVecU* pVars, cuVecU* units, cuVecU* resolved)
		{
			uint32 tx = threadIdx.x;
			uint32 tid = global_tx();
			__shared__ uint32 defs[BLVE * FAN_LMT];
			__shared__ uint32 outs[BLVE * SH_MAX_BVE_OUT];
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				uint32 p = V2D(x), n = NEG(p);
				uint32 pOrgs = (*ot)[p].size(), nOrgs = (*ot)[n].size();
				// try pure-literal elimination
				if (!pOrgs || !nOrgs) {
					uint32 psLits = 0, nsLits = 0;
					countLitsBefore(*cnf, (*ot)[p], psLits);
					countLitsBefore(*cnf, (*ot)[n], nsLits);
					toblivion(*cnf, (*ot)[p], (*ot)[n], resolved, pOrgs, nOrgs, psLits, nsLits, p);
					x |= MELTING_MASK;
				}
				// try simple resolution
				else if ((pOrgs == 1 || nOrgs == 1) &&
					resolve_x(x, pOrgs, nOrgs, *cnf, (*ot)[p], (*ot)[n], units, resolved, &outs[tx * SH_MAX_BVE_OUT])) x |= MELTING_MASK;
				// try gate-equivalence reasoning, otherwise resolution
				else if (gateReasoning_x(p, pOrgs, nOrgs, *cnf, (*ot)[p], (*ot)[n], units, resolved, &defs[tx * FAN_LMT], &outs[tx * SH_MAX_BVE_OUT]) ||
					resolve_x(x, pOrgs, nOrgs, *cnf, (*ot)[p], (*ot)[n], units, resolved, &outs[tx * SH_MAX_BVE_OUT])) x |= MELTING_MASK;
				tid += stride_x();
			}
		}

		__global__ void in_ve_k(CNF* cnf, OT* ot, cuVecU* pVars, cuVecU* units, cuVecU* resolved)
		{
			uint32 tx = threadIdx.x;
			uint32 tid = global_tx();
			__shared__ uint32 defs[BLVE * FAN_LMT];
			__shared__ uint32 outs[BLVE * SH_MAX_BVE_OUT];
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				uint32 p = V2D(x), n = NEG(p);
				uint32 pOrgs = 0, nOrgs = 0;
				countOrgs(*cnf, (*ot)[p], pOrgs), countOrgs(*cnf, (*ot)[n], nOrgs);
				// try pure-literal elimination
				if (!pOrgs || !nOrgs) {
					uint32 psLits = 0, nsLits = 0;
					countLitsBefore(*cnf, (*ot)[p], psLits);
					countLitsBefore(*cnf, (*ot)[n], nsLits);
					toblivion(*cnf, (*ot)[p], (*ot)[n], resolved, pOrgs, nOrgs, psLits, nsLits, p);
					x |= MELTING_MASK;
				}
				// try simple resolution
				else if ((pOrgs == 1 || nOrgs == 1) &&
					resolve_x(x, pOrgs, nOrgs, *cnf, (*ot)[p], (*ot)[n], units, resolved, &outs[tx * SH_MAX_BVE_OUT])) x |= MELTING_MASK;
				// try gate-equivalence reasoning, otherwise resolution
				else if (gateReasoning_x(p, pOrgs, nOrgs, *cnf, (*ot)[p], (*ot)[n], units, resolved, &defs[tx * FAN_LMT], &outs[tx * SH_MAX_BVE_OUT]) ||
					resolve_x(x, pOrgs, nOrgs , *cnf, (*ot)[p], (*ot)[n], units, resolved, &outs[tx * SH_MAX_BVE_OUT])) x |= MELTING_MASK;
				tid += stride_x();
			}
		}

		__global__ void hse_k(CNF* cnf, OT* ot, cuVecU* pVars, cuVecU* units, uint32 limit)
		{
			uint32 tid = global_tx();
			__shared__ uint32 sh_cls[BLHSE * SH_MAX_HSE_IN];
			while (tid < pVars->size()) {
				uint32 x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				uint32 p = V2D(x), n = NEG(p);
				if ((*ot)[p].size() <= limit && (*ot)[n].size() <= limit)
					self_sub_x(p, *cnf, (*ot)[p], (*ot)[n], units, &sh_cls[threadIdx.x * SH_MAX_HSE_IN]);
				tid += stride_x();
			}
		}

		__global__ void bce_k(CNF* cnf, OT* ot, cuVecU* pVars, uint32 limit)
		{
			uint32 tid = global_tx();
			__shared__ uint32 sh_cls[BLBCE * SH_MAX_BCE_IN];
			while (tid < pVars->size()) {
				uint32 x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				uint32 p = V2D(x), n = NEG(p);
				if ((*ot)[p].size() <= limit && (*ot)[n].size() <= limit)
					blocked_x(x, *cnf, (*ot)[p], (*ot)[n], &sh_cls[threadIdx.x * SH_MAX_BCE_IN]);
				tid += stride_x();
			}
		}

		__global__ void hre_k(CNF* cnf, OT* ot, cuVecU* pVars, uint32 limit)
		{
			uint32 gid = global_ty();
			uint32* smem = SharedMemory<uint32>();
			uint32* m_c = smem + warpSize * SH_MAX_HRE_IN + threadIdx.y * SH_MAX_HRE_OUT; // shared memory for resolvent
			while (gid < pVars->size()) {
				assert(pVars->at(gid));
				assert(!ELIMINATED(pVars->at(gid)));
				uint32 p = V2D(pVars->at(gid)), n = NEG(p);
				// do merging and apply forward equality check (on-the-fly) over resolvents
				if ((*ot)[p].size() <= limit && (*ot)[n].size() <= limit) {
					for (uint32* i = (*ot)[p]; i != (*ot)[p].end(); i++) {
						SCLAUSE& pos = (*cnf)[*i];
						if (pos.deleted() || pos.learnt()) continue;
						if (pos.size() <= SH_MAX_HRE_IN) { // use shared memory for positives
							uint32* sh_pos = smem + threadIdx.y * SH_MAX_HRE_IN;
							if (threadIdx.x == 0) pos.shareTo(sh_pos);
							for (uint32* j = (*ot)[n]; j != (*ot)[n].end(); j++) {
								SCLAUSE& neg = (*cnf)[*j];
								if (neg.deleted() || neg.learnt() || (pos.size() + neg.size() - 2) > SH_MAX_HRE_OUT) continue;
								int m_len = 0;
								if (threadIdx.x == 0) m_len = merge(pVars->at(gid), sh_pos, pos.size(), neg, m_c);
								m_len = __shfl_sync(FULLWARP, m_len, 0);
								if (m_len) forward_equ(*cnf, *ot, m_c, m_len);
							}
						}
						else { // use global memory
							for (uint32* j = (*ot)[n]; j != (*ot)[n].end(); j++) {
								SCLAUSE& neg = (*cnf)[*j];
								if (neg.deleted() || neg.learnt() || (pos.size() + neg.size() - 2) > SH_MAX_HRE_OUT) continue;
								int m_len = 0;
								if (threadIdx.x == 0) m_len = merge(pVars->at(gid), pos, neg, m_c);
								m_len = __shfl_sync(FULLWARP, m_len, 0);
								if (m_len) forward_equ(*cnf, *ot, m_c, m_len);
							}
						}
					}
				}
				gid += stride_y();
			}
		}
		//==============================================//
		//          ParaFROST Wrappers/helpers          //
		//==============================================//
		void cuMemSetAsync(addr_t mem, const Byte& val, const size_t& size)
		{
			uint32 nBlocks = MIN(uint32((size + BLOCK1D - 1) / BLOCK1D), maxGPUThreads / BLOCK1D);
			memset_k<Byte> << <nBlocks, BLOCK1D >> > (mem, val, size);
		}
		void copyIf(uint32* dest, CNF* src, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			uint32 nBlocks = MIN((inf.nClauses + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			copy_if_k << <nBlocks, BLOCK1D >> > (dest, src, gstats);
			LOGERR("Copying literals failed");
			CHECK(hipDeviceSynchronize());
		}
		void shrinkSimp(CNF* dest, CNF* src)
		{
			uint32 cnf_size = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 nBlocks = MIN((cnf_size + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			copy_if_k << <nBlocks, BLOCK1D >> > (dest, src);
			LOGERR("Copying CNF failed");
			CHECK(hipDeviceSynchronize());
		}
		void calcScores(VARS* vars, uint32* hist)
		{
			uint32 nBlocks = MIN((inf.maxVar + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			assign_scores << <nBlocks, BLOCK1D >> > (vars->eligible, vars->scores, hist, inf.maxVar);
			LOGERR("Assigning scores failed");
			CHECK(hipDeviceSynchronize());
		}
		void calcScores(VARS* vars, uint32* hist, OT* ot)
		{
			uint32 nBlocks = MIN((inf.maxVar + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			assign_scores << <nBlocks, BLOCK1D >> > (vars->eligible, vars->scores, hist, ot, inf.maxVar);
			LOGERR("Assigning scores failed");
			CHECK(hipDeviceSynchronize());
		}
		void calcSigCNFAsync(CNF* cnf, const uint32& offset, const uint32& size, const hipStream_t& _s)
		{
			assert(size);
			uint32 nBlocks = MIN((size + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			calc_sig_k << <nBlocks, BLOCK1D, 0, _s >> > (cnf, offset, size);
		}
		void calcSigCNF(CNF* cnf, const uint32& size)
		{
			assert(size);
			uint32 nBlocks = MIN((size + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			calc_sig_k << <nBlocks, BLOCK1D >> > (cnf, 0, size);
			LOGERR("Signature calculation failed");
			CHECK(hipDeviceSynchronize());
		}
		void reduceOTAsync(CNF* cnf, OT* ot, const bool& p)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			uint32 nBlocks = MIN(uint32((inf.nDualVars + BLOCK1D - 1) / BLOCK1D), maxGPUThreads / BLOCK1D);
			reduce_ot << <nBlocks, BLOCK1D >> > (cnf, ot);
			if (p) {
				LOGERR("Occurrence table reduction failed");
				CHECK(hipDeviceSynchronize());
				PFLOGR('=', 30);
				PFLOG0("\toccurrence table");
				ot->print();
				PFLOGR('=', 30);
			}
		}
		void reduceOT(CNF* cnf, OT* ot, VARS* vars, hipStream_t* streams, const bool& p)
		{
			assert(vars->numPVs);
			assert(cnf != NULL);
			assert(ot != NULL);
			uint32 nBlocks = MIN((vars->numPVs + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			reduce_ot_p << <nBlocks, BLOCK1D, 0, streams[0] >> > (cnf, ot, vars->pVars);
			reduce_ot_n << <nBlocks, BLOCK1D, 0, streams[1] >> > (cnf, ot, vars->pVars);
			LOGERR("Occurrence table reduction failed");
			CHECK(hipDeviceSynchronize());
			assert(ot->accViolation());
			if (p) {	
				PFLOGR('=', 30);
				PFLOG0("\toccurrence table");
				ot->print();
				PFLOGR('=', 30);
			}
		}
		void createOT(CNF* cnf, OT* ot, const bool& p)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			uint32 cnf_size = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 rstGridSize = MIN(uint32((inf.nDualVars + BLOCK1D - 1) / BLOCK1D), maxGPUThreads / BLOCK1D);
			uint32 otGridSize = MIN((cnf_size + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			reset_ot_k << <rstGridSize, BLOCK1D >> > (ot);
			create_ot_k << <otGridSize, BLOCK1D >> > (cnf, ot);
			LOGERR("Occurrence table creation failed");
			CHECK(hipDeviceSynchronize());
			assert(ot->accViolation());
			if (p) {
				PFLOGR('=', 30);
				PFLOG0("\toccurrence table");
				ot->print();
				PFLOGR('=', 30);
			}
		}
		void createOTAsync(CNF* cnf, OT* ot, const bool& p)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			uint32 cnf_size = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 rstGridSize = MIN(uint32((inf.nDualVars + BLOCK1D - 1) / BLOCK1D), maxGPUThreads / BLOCK1D);
			uint32 otGridSize = MIN((cnf_size + BLOCK1D - 1) / BLOCK1D, maxGPUThreads / BLOCK1D);
			reset_ot_k << <rstGridSize, BLOCK1D >> > (ot);
			create_ot_k << <otGridSize, BLOCK1D >> > (cnf, ot);
			if (p) {
				LOGERR("Occurrence table creation failed");
				CHECK(hipDeviceSynchronize());
				assert(ot->accViolation());
				PFLOGR('=', 30);
				PFLOG0("\toccurrence table");
				ot->print();
				PFLOGR('=', 30);
			}
		}
		void ve(CNF* cnf, OT* ot, VARS* vars, const bool& in)
		{
			assert(vars->numPVs);
#if VE_DBG
			putchar('\n');
			if (in) in_ve_k << <1, 1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved);
			else	   ve_k << <1, 1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved);
#else
			uint32 nBlocks = MIN((vars->numPVs + BLVE - 1) / BLVE, maxGPUThreads / BLVE);
			if (in)	in_ve_k << <nBlocks, BLVE >> > (cnf, ot, vars->pVars, vars->units, vars->resolved);
			else	   ve_k << <nBlocks, BLVE >> > (cnf, ot, vars->pVars, vars->units, vars->resolved);
#endif
			LOGERR("Parallel BVE failed");
			CHECK(hipDeviceSynchronize());
		}
		void hse(CNF* cnf, OT* ot, VARS* vars, const uint32& limit)
		{
			assert(vars->numPVs);
			assert(limit);
#if SS_DBG
			putchar('\n');
			hse_k << <1, 1 >> > (cnf, ot, vars->pVars, vars->units, limit);
#else
			uint32 nBlocks = MIN((vars->numPVs + BLHSE - 1) / BLHSE, maxGPUThreads / BLHSE);
			hse_k << <nBlocks, BLHSE >> > (cnf, ot, vars->pVars, vars->units, limit);
#endif
			LOGERR("Parallel HSE failed");
			CHECK(hipDeviceSynchronize());
		}
		void bce(CNF* cnf, OT* ot, VARS* vars, const uint32& limit)
		{
			assert(vars->numPVs);
			assert(limit);
			uint32 nBlocks = MIN((vars->numPVs + BLBCE - 1) / BLBCE, maxGPUThreads / BLBCE);
			bce_k << <nBlocks, BLBCE >> > (cnf, ot, vars->pVars, limit);
			LOGERR("Parallel BCE failed");
			CHECK(hipDeviceSynchronize());
		}
		void hre(CNF* cnf, OT* ot, VARS* vars, const uint32& limit)
		{
			assert(vars->numPVs);
			assert(limit);
			dim3 block2D(devProp.warpSize, devProp.warpSize), grid2D(1, 1, 1);
			grid2D.y = MIN((vars->numPVs + block2D.y - 1) / block2D.y, maxGPUThreads / block2D.y);
			uint32 smemSize = devProp.warpSize * (SH_MAX_HRE_IN + SH_MAX_HRE_OUT) * sizeof(uint32);
			hre_k << <grid2D, block2D, smemSize >> > (cnf, ot, vars->pVars, limit);
			LOGERR("HRE Elimination failed");
			CHECK(hipDeviceSynchronize());
		}
		void countMelted(LIT_ST* vstate)
		{
			inf.n_del_vars_after = 0;
			for (uint32 v = 1; v <= inf.maxVar; v++) 
				if (vstate[v] == MELTED)
					inf.n_del_vars_after++;
			assert(inf.n_del_vars_after >= inf.maxMelted);
			inf.n_del_vars_after -= inf.maxMelted;
			inf.maxMelted += inf.n_del_vars_after;
		}
		void evalReds(CNF* cnf, GSTATS* gstats, LIT_ST* vstate)
		{
			reset_stats << <1, 1 >> > (gstats);
			uint32 cnf_sz = inf.nClauses + (inf.maxAddedCls >> 1); // approximate cnf size 
			uint32 nBlocks1 = MIN((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUThreads / (BLOCK1D << 1));
			uint32 smemSize1 = BLOCK1D * sizeof(uint32) * 2;
			cnt_reds << <nBlocks1, BLOCK1D, smemSize1 >> > (cnf, gstats);
			countMelted(vstate);
			LOGERR("Counting reductions failed");
			CHECK(hipDeviceSynchronize());
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost)); // avoids unified memory migration on large scale
			inf.n_cls_after = hstats.numClauses;
			inf.n_lits_after = hstats.numLits;
		}
		void countFinal(CNF* cnf, GSTATS* gstats, LIT_ST* vstate)
		{
			reset_stats << <1, 1 >> > (gstats);
			uint32 cnf_sz = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 nBlocks = MIN((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUThreads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * sizeof(uint32);
			cnt_cls << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			countMelted(vstate);
			LOGERR("Counting clauses failed");
			CHECK(hipDeviceSynchronize());
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
		}
		void countCls(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1>> > (gstats);
			uint32 cnf_sz = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 nBlocks = MIN((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUThreads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * sizeof(uint32);
			cnt_cls << <nBlocks, BLOCK1D, smemSize>> > (cnf, gstats);
			LOGERR("Counting clauses failed");
			CHECK(hipDeviceSynchronize());
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
		}
		void countLits(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			uint32 cnf_sz = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 nBlocks = MIN((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUThreads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * sizeof(uint32);
			cnt_lits << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			LOGERR("Counting literals failed");
			CHECK(hipDeviceSynchronize());
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_lits_after = hstats.numLits;
		}
		void countAll(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			uint32 cnf_sz = inf.nClauses + (inf.maxAddedCls >> 1);
			uint32 nBlocks = MIN((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUThreads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * (sizeof(uint32) + sizeof(uint32));
			cnt_cls_lits << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			LOGERR("Counting clauses-literals failed");
			CHECK(hipDeviceSynchronize());
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
			inf.n_lits_after = hstats.numLits;
		}

	}
}