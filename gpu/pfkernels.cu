#include "hip/hip_runtime.h"
/***********************************************************************[pfkernels.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "pfsimplify.cuh"
#include "pfmemory.cuh"
#include "pfdevice.cuh"
#include "pfbve.cuh"
#include "pfhse.cuh"
#include "pfere.cuh"
#include <cub/device/device_scan.cuh>

using namespace hipcub;

namespace pFROST {

	namespace SIGmA {

		template<class T>
		__global__ void memset_k(T* mem, T val, size_t size)
		{
			size_t tid = global_tx();
			while (tid < size) { mem[tid] = val; tid += stride_x(); }
		}

		__global__ void reset_stats(GSTATS* gstats) { gstats->numDelVars = 0, gstats->numClauses = 0, gstats->numLits = 0; }

		__global__ void prep_cnf_k(CNF* cnf)
		{
			uint32 tid = global_tx();
			while (tid < cnf->size()) { 
				SCLAUSE& c = cnf->clause(tid);
				devSort(c.data(), c.size());
				calcSig(c); 
				tid += stride_x();
			}
		}

		__global__ void reset_ot_k(OT* ot)
		{
			uint32 tid = global_tx();
			while (tid < ot->size()) { (*ot)[tid].clear(); tid += stride_x(); }
		}

		__global__ void reduce_ot(const CNF* __restrict__ cnf, OT* __restrict__ ot)
		{
			uint32 tid = global_tx();
			while (tid < ot->size()) { reduceOL(*cnf, (*ot)[tid]); tid += stride_x(); }
		}

		__global__ void sort_ot_p(const CNF* __restrict__ cnf, OT* __restrict__ ot, const cuVecU* __restrict__ pVars)
		{
			uint32 tid = global_tx();
			while (tid < pVars->size()) {
				const uint32 x = pVars->at(tid), p = V2L(x);
				assert(x);
				OL& ol = (*ot)[p];
				devSort(ol.data(), ol.size(), CNF_CMP_KEY(cnf));
				tid += stride_x();
			}
		}

		__global__ void sort_ot_n(const CNF* __restrict__ cnf, OT* __restrict__ ot, const cuVecU* __restrict__ pVars)
		{
			uint32 tid = global_tx();
			while (tid < pVars->size()) {
				const uint32 x = pVars->at(tid), n = NEG(V2L(x));
				assert(x);
				OL& ol = (*ot)[n];
				devSort(ol.data(), ol.size(), CNF_CMP_KEY(cnf));
				tid += stride_x();
			}
		}

		__global__ void create_ot_k(CNF* __restrict__ cnf, OT* __restrict__ ot)
		{
			uint32 tid = global_tx();
			while (tid < cnf->size()) {
				const S_REF r = cnf->ref(tid);
				SCLAUSE& c = (*cnf)[r];
				if (c.original() || c.learnt()) {
#pragma unroll
					forall_clause(c, lit) (*ot)[*lit].insert(r);
				}
				tid += stride_x();
			}
		}

		__global__ void assign_scores(uint32* __restrict__ eligible, uint32* __restrict__ scores, const uint32* __restrict__ hist, uint32 size)
		{
			uint32 tid = global_tx();
			while (tid < size) {
				const uint32 v = tid + 1;
				const uint32 p = V2L(v), ps = hist[p], ns = hist[NEG(p)];
				eligible[tid] = v;
				scores[v] = ps * ns;
				tid += stride_x();
			}
		}

		__global__ void assign_scores(uint32* __restrict__ eligible, uint32* __restrict__ scores, uint32* __restrict__ hist, const OT* __restrict__ ot, uint32 size)
		{
			uint32 tid = global_tx();
			while (tid < size) {
				const uint32 v = tid + 1;
				const uint32 p = V2L(v), n = NEG(p), ps = (*ot)[p].size(), ns = (*ot)[n].size();
				hist[p] = ps, hist[n] = ns;
				eligible[tid] = v;
				scores[v] = ps * ns;
				tid += stride_x();
			}
		}

		__global__ void copy_if_k(uint32* __restrict__ dest, CNF* __restrict__ src, GSTATS* __restrict__ gstats)
		{
			uint32 tid = global_tx();
			while (tid < src->size()) {
				SCLAUSE& c = src->clause(tid);
				if (c.original() || c.learnt()) {
					uint32* d = dest + atomicAdd(&gstats->numLits, c.size());
#pragma unroll
					forall_clause(c, s) { *d++ = *s; }
				}
				tid += stride_x();
			}
		}

		__global__ void cnt_reds(const CNF* __restrict__ cnf, GSTATS* __restrict__ gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32* sh_rLits = sh_rCls + blockDim.x;
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				const SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt())
					nCls++, nLits += c1.size();
				if (tid + blockDim.x < cnf->size()) {
					const SCLAUSE& c2 = cnf->clause(tid + blockDim.x);
					if (c2.original() || c2.learnt())
						nCls++, nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rCls, nCls, sh_rLits, nLits);
			warpReduce(sh_rCls, nCls, sh_rLits, nLits);
			if (threadIdx.x == 0) {
				atomicAdd(&gstats->numClauses, nCls);
				atomicAdd(&gstats->numLits, nLits);
			}
		}

		__global__ void cnt_cls(const CNF* __restrict__ cnf, GSTATS* __restrict__ gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			while (tid < cnf->size()) {
				const SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nCls++;
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					const SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nCls++;
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, cnf->size());
			sharedReduce(sh_rCls, nCls);
			warpReduce(sh_rCls, nCls);
			if (threadIdx.x == 0) atomicAdd(&gstats->numClauses, nCls);
		}

		__global__ void cnt_lits(const CNF* __restrict__ cnf, GSTATS* __restrict__ gstats)
		{
			uint32* sh_rLits = SharedMemory<uint32>();
			uint32 tid = global_tx_off();
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				const SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nLits += c1.size();
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					const SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rLits, nLits);
			warpReduce(sh_rLits, nLits);
			if (threadIdx.x == 0) atomicAdd(&gstats->numLits, nLits);
		}

		__global__ void cnt_cls_lits(const CNF* __restrict__ cnf, GSTATS* __restrict__ gstats)
		{
			uint32* sh_rCls = SharedMemory<uint32>();
			uint32* sh_rLits = sh_rCls + blockDim.x;
			uint32 tid = global_tx_off();
			uint32 nCls = 0;
			uint32 nLits = 0;
			while (tid < cnf->size()) {
				const SCLAUSE& c1 = cnf->clause(tid);
				if (c1.original() || c1.learnt()) nCls++, nLits += c1.size();
				uint32 off = tid + blockDim.x;
				if (off < cnf->size()) {
					const SCLAUSE& c2 = cnf->clause(off);
					if (c2.original() || c2.learnt()) nCls++, nLits += c2.size();
				}
				tid += stride_x_off();
			}
			loadShared(sh_rCls, nCls, sh_rLits, nLits, cnf->size());
			sharedReduce(sh_rCls, nCls, sh_rLits, nLits);
			warpReduce(sh_rCls, nCls, sh_rLits, nLits);
			if (threadIdx.x == 0) {
				atomicAdd(&gstats->numClauses, nCls);
				atomicAdd(&gstats->numLits, nLits);
			}
		}

		__global__ void ve_k(CNF* __restrict__ cnfptr, OT* __restrict__ otptr, cuVecU* __restrict__ pVars, cuVecU* __restrict__ units, cuVecU* __restrict__ resolved, const uint32* __restrict__ vorg)
		{
			uint32 tid = global_tx();
			__shared__ uint32 outs[BLVE * SH_MAX_BVE_OUT];
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				const uint32 p = V2L(x), n = NEG(p);
				CNF& cnf = *cnfptr;
				OT& ot = *otptr;
				OL& poss = ot[p], &negs = ot[n];
				uint32 pOrgs = 0, nOrgs = 0;
				countOrgs(cnf, poss, pOrgs), countOrgs(cnf, negs, nOrgs);
				bool elim = false;
				// pure-literal elimination
				if (!pOrgs || !nOrgs) {
					toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved), elim = true;
				}
				// Equiv/NOT-gate Reasoning
				else if (uint32 def = find_equ_gate(p, cnf, poss, negs)) {
					saveResolved(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved); // must be called before substitution
					substitute_single(p, def, cnf, poss, negs, units), elim = true;
				}
				else {
					assert(pOrgs && nOrgs);
					const uint32 nClsBefore = pOrgs + nOrgs;
					uint32 *shared_outs = outs + threadIdx.x * SH_MAX_BVE_OUT;
					uint32 nAddedCls, nAddedLits;
					// AND-gate Reasoning
					if (nOrgs < SH_MAX_BVE_OUT && find_ao_gate(n, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits)) {
						if (nAddedCls) substitute_x(x, nAddedCls, nAddedLits, cnf, poss, negs, units, shared_outs);
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved), elim = true;
					}
					// OR-gate Reasoning
					else if (pOrgs < SH_MAX_BVE_OUT && find_ao_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits)) {
						if (nAddedCls) substitute_x(x, nAddedCls, nAddedLits, cnf, poss, negs, units, shared_outs);
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved), elim = true;
					}
					// ITE-gate Reasoning
					else if (find_ite_gate(p, nClsBefore, cnf, ot, nAddedCls, nAddedLits)
						|| find_ite_gate(n, nClsBefore, cnf, ot, nAddedCls, nAddedLits)) {
						if (nAddedCls) substitute_x(x, nAddedCls, nAddedLits, cnf, poss, negs, units, shared_outs);
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved), elim = true;
					}
					// XOR-gate Reasoning
					else if (find_xor_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits)) {
						if (nAddedCls) substitute_x(x, nAddedCls, nAddedLits, cnf, poss, negs, units, shared_outs);
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved), elim = true;
					}
					// n-by-m resolution
					else if (resolve(x, nClsBefore, cnf, poss, negs, nAddedCls, nAddedLits)) {
						if (nAddedCls) resolve_x(x, nAddedCls, nAddedLits, cnf, poss, negs, units, shared_outs);
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved);
						elim = true;
					}
				}
				if (elim) x |= MELTING_MASK;
				tid += stride_x();
			}
		}

		__global__ void in_ve_k_1(CNF* __restrict__ cnfptr, OT* __restrict__ otptr, cuVecU* __restrict__ pVars, cuVecU* __restrict__ units, cuVecU* __restrict__ resolved, const uint32* __restrict__ vorg, uint32* __restrict__ type, uint32* __restrict__ rpos, S_REF* __restrict__ rref)
		{
			uint32 tid = global_tx();
			uint32* outs = SharedMemory<uint32>();
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				const uint32 p = V2L(x), n = NEG(p);
				CNF& cnf = *cnfptr;
				OT& ot = *otptr;
				OL& poss = ot[p], & negs = ot[n];
				uint32 pOrgs = 0, nOrgs = 0;
				countOrgs(cnf, poss, pOrgs), countOrgs(cnf, negs, nOrgs);
				// pure-literal elimination
				if (!pOrgs || !nOrgs) {
					toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved);
					type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
				}
				// Equiv/NOT-gate Reasoning
				else if (uint32 def = find_equ_gate(p, cnf, poss, negs)) {
					saveResolved(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved); // must be called before substitution
					substitute_single(p, def, cnf, poss, negs, units);
					type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
				}
				else {
					assert(pOrgs && nOrgs);
					const uint32 nClsBefore = pOrgs + nOrgs;
					uint32* shared_outs = outs + threadIdx.x * SH_MAX_BVE_OUT1;
					uint32 elimType = 0, nAddedCls = 0, nAddedLits = 0;
					//=====================
					// check resolvability 
					//=====================
					// AND/OR-gate Reasoning
					if ((nOrgs < SH_MAX_BVE_OUT1 && find_ao_gate(n, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits))
					||  (pOrgs < SH_MAX_BVE_OUT1 && find_ao_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits))) elimType = AOIX_MASK;
					// ITE-gate Reasoning
					else if (find_ite_gate(p, nClsBefore, cnf, ot, nAddedCls, nAddedLits)
						||	 find_ite_gate(n, nClsBefore, cnf, ot, nAddedCls, nAddedLits)) elimType = AOIX_MASK;
					// XOR-gate Reasoning
					else if (find_xor_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits)) elimType = AOIX_MASK;
					// n-by-m resolution
					else if (!nAddedCls && resolve(x, nClsBefore, cnf, poss, negs, nAddedCls, nAddedLits)) elimType = RES_MASK;
					//=====================
					// check addibility 
					//=====================
					if (!nAddedCls) { // eliminated without resolvents
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved);
						type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
					}
					else if (elimType) { // can be eliminated with resolvents in next phase
						assert(nAddedLits >= nAddedCls);
						assert(elimType < TYPE_MASK);
						assert(nAddedCls <= ADDEDCLS_MAX);
						assert(nAddedLits <= ADDEDLITS_MAX);
						// save elimination info.
						type[tid] = ENCODEVARINFO(elimType, nAddedCls, nAddedLits);
						rpos[tid] = nAddedCls, rref[tid] = nAddedLits + dc_nbuckets * nAddedCls;			
					}
					else  // cannot be eliminated
						type[tid] = 0, rref[tid] = 0, rpos[tid] = 0;
				}
				tid += stride_x();
			}
		}

		__global__ void ve_k_1(CNF* __restrict__ cnfptr, OT* __restrict__ otptr, cuVecU* __restrict__ pVars, cuVecU* __restrict__ units, cuVecU* __restrict__ resolved, const uint32* __restrict__ vorg, uint32* __restrict__ type, uint32* __restrict__ rpos, S_REF* __restrict__ rref)
		{
			uint32 tid = global_tx();
			uint32* outs = SharedMemory<uint32>();
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				const uint32 p = V2L(x), n = NEG(p);
				CNF& cnf = *cnfptr;
				OT& ot = *otptr;
				OL& poss = ot[p], & negs = ot[n];
				const uint32 pOrgs = poss.size(), nOrgs = negs.size();
				// pure-literal elimination
				if (!pOrgs || !nOrgs) {
					toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved);
					type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
				}
				// Equiv/NOT-gate Reasoning
				else if (uint32 def = find_equ_gate(p, cnf, poss, negs)) {
					saveResolved(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved); // must be called before substitution
					substitute_single(p, def, cnf, poss, negs, units);
					type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
				}
				else {
					assert(pOrgs && nOrgs);
					const uint32 nClsBefore = pOrgs + nOrgs;
					uint32* shared_outs = outs + threadIdx.x * SH_MAX_BVE_OUT1;
					uint32 elimType = 0, nAddedCls = 0, nAddedLits = 0;
					//=====================
					// check resolvability 
					//=====================
					// AND/OR-gate Reasoning
					if ((nOrgs < SH_MAX_BVE_OUT1 && find_ao_gate(n, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits))
					||  (pOrgs < SH_MAX_BVE_OUT1 && find_ao_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits))) elimType = AOIX_MASK;
					// ITE-gate Reasoning
					else if (find_ite_gate(p, nClsBefore, cnf, ot, nAddedCls, nAddedLits) 
						  || find_ite_gate(n, nClsBefore, cnf, ot, nAddedCls, nAddedLits)) elimType = AOIX_MASK;
					// XOR-gate Reasoning
					else if (find_xor_gate(p, nClsBefore, cnf, ot, shared_outs, nAddedCls, nAddedLits)) elimType = AOIX_MASK;
					// n-by-m resolution
					else if (!nAddedCls && resolve(x, nClsBefore, cnf, poss, negs, nAddedCls, nAddedLits)) elimType = RES_MASK;
					//=====================
					// check addibility 
					//=====================
					if (!nAddedCls) { // eliminated without resolvents
						toblivion(p, vorg, pOrgs, nOrgs, cnf, poss, negs, resolved);
						type[tid] = 0, rref[tid] = 0, rpos[tid] = 0, x |= MELTING_MASK;
					}
					else if (elimType) { // can be eliminated with resolvents in next phase
						assert(nAddedLits >= nAddedCls);
						assert(elimType < TYPE_MASK);
						assert(nAddedCls <= ADDEDCLS_MAX);
						assert(nAddedLits <= ADDEDLITS_MAX);
						// save elimination info.
						type[tid] = ENCODEVARINFO(elimType, nAddedCls, nAddedLits);
						rpos[tid] = nAddedCls, rref[tid] = nAddedLits + dc_nbuckets * nAddedCls;
					}
					else  // cannot be eliminated
						type[tid] = 0, rref[tid] = 0, rpos[tid] = 0;
				}
				tid += stride_x();
			}
		}

		__global__ void ve_k_2(CNF* __restrict__ cnf, OT* __restrict__ ot, cuVecU* __restrict__ pVars, cuVecU* __restrict__ units, cuVecU* __restrict__ resolved, const uint32* __restrict__ vorg, const uint32* __restrict__ type, const uint32* __restrict__ rpos, const S_REF* __restrict__ rref)
		{
			uint32 tid = global_tx();
			uint32* outs = SharedMemory<uint32>();
			while (tid < pVars->size()) {
				uint32& x = (*pVars)[tid];
				assert(x);
				const uint32 xinfo = type[tid];
				const uint32 elimType = RECOVERTYPE(xinfo);
				assert(elimType < TYPE_MASK);
				if (elimType) {
					assert(!ELIMINATED(x));
					const uint32 p = V2L(x);
					const uint32 nAddedCls = RECOVERADDEDCLS(xinfo);
					const uint32 nAddedLits = RECOVERADDEDLITS(xinfo);
					assert(nAddedCls && nAddedCls <= ADDEDCLS_MAX);
					assert(nAddedLits && nAddedLits <= ADDEDLITS_MAX);
					const uint32 added_pos = rpos[tid];
					const S_REF added_ref = rref[tid];
					OL& poss = (*ot)[p], &negs = (*ot)[NEG(p)];
					if (IS_RES(elimType)) {
						if (memorySafe(tid, x, nAddedCls, nAddedLits, added_pos, added_ref, cnf)) {
							if (nAddedCls) saveResolved(p, vorg, *cnf, poss, negs, resolved);
							resolve_x(x, nAddedCls, nAddedLits, added_pos, added_ref, *cnf, poss, negs, units, outs + threadIdx.x * SH_MAX_BVE_OUT2);
							x |= MELTING_MASK;
						}
					}
					else {
						assert(IS_AOIX(elimType));
						if (memorySafe(tid, x, nAddedCls, nAddedLits, added_pos, added_ref, cnf)) {
							if (nAddedCls) saveResolved(p, vorg, *cnf, poss, negs, resolved);
							substitute_x(x, nAddedCls, nAddedLits, added_pos, added_ref, *cnf, poss, negs, units, outs + threadIdx.x * SH_MAX_BVE_OUT2);
							x |= MELTING_MASK;
						}
						else freezeClauses(*cnf, poss, negs);
					}
				}
				tid += stride_x();
			}
		}

		__global__ void roll_back_gates(CNF* __restrict__ cnf, OT* __restrict__ ot, cuVecU* __restrict__ pVars, uint32* __restrict__ type)
		{
			uint32 tid = global_tx();
			while (tid < pVars->size()) {
				const uint32 x = (*pVars)[tid];
				assert(x);
				const uint32 xinfo = type[tid];
				const uint32 elimType = RECOVERTYPE(xinfo);
				assert(elimType < TYPE_MASK);
				if (elimType && IS_AOIX(elimType)) {
					assert(!ELIMINATED(x));
					const uint32 p = V2L(x);
					freezeClauses(*cnf, (*ot)[p], (*ot)[NEG(p)]);
				}
				tid += stride_x();
			}
		}

		__global__ void hse_k(CNF* __restrict__ cnf, OT* __restrict__ ot, const cuVecU* __restrict__ pVars, cuVecU* __restrict__ units)
		{
			uint32 tid = global_tx();
			__shared__ uint32 sh_cls[BLHSE * SH_MAX_HSE_IN];
			while (tid < pVars->size()) {
				const uint32 x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				const uint32 p = V2L(x), n = NEG(p);
				if ((*ot)[p].size() <= dc_limits[0] && (*ot)[n].size() <= dc_limits[0])
					subsume_x(p, *cnf, (*ot)[p], (*ot)[n], units, sh_cls + threadIdx.x * SH_MAX_HSE_IN);
				tid += stride_x();
			}
		}

		__global__ void bce_k(CNF* __restrict__ cnf, OT* __restrict__ ot, const cuVecU* __restrict__ pVars, cuVecU* __restrict__ resolved, const uint32* __restrict__ vorg)
		{
			uint32 tid = global_tx();
			__shared__ uint32 sh_cls[BLBCE * SH_MAX_BCE_IN];
			while (tid < pVars->size()) {
				const uint32 x = (*pVars)[tid];
				assert(x);
				assert(!ELIMINATED(x));
				const uint32 p = V2L(x), n = NEG(p);
				if ((*ot)[p].size() <= dc_limits[1] && (*ot)[n].size() <= dc_limits[1])
					blocked_x(x, vorg, *cnf, (*ot)[p], (*ot)[n], resolved, sh_cls + threadIdx.x * SH_MAX_BCE_IN);
				tid += stride_x();
			}
		}

		__global__ void ere_k(CNF* __restrict__ cnf, OT* __restrict__ ot, const cuVecU* __restrict__ pVars)
		{
			uint32 gid = global_ty();
			uint32* smem = SharedMemory<uint32>();
			while (gid < pVars->size()) {
				const uint32 v = pVars->at(gid);
				assert(v);
				assert(!ELIMINATED(v));
				const uint32 p = V2L(v), n = NEG(p);
				OL& poss = (*ot)[p], & negs = (*ot)[n];
				// do merging and apply forward equality check (on-the-fly) over resolvents
				if (poss.size() <= dc_limits[2] && negs.size() <= dc_limits[2]) {
					forall_occurs(poss, i) {
						SCLAUSE& pos = (*cnf)[*i];
						if (pos.deleted()) continue;
						forall_occurs(negs, j) {
							SCLAUSE& neg = (*cnf)[*j];
							if (neg.deleted() || (pos.size() + neg.size() - 2) > SH_MAX_ERE_OUT) continue;
							uint32* m_c = smem + threadIdx.y * SH_MAX_ERE_OUT; // shared memory for resolvent
							int m_len = 0;
							if ((m_len = merge_ere(v, pos, neg, m_c)) > 1) {
								CL_ST type;
								if (pos.learnt() || neg.learnt()) type = LEARNT;
								else type = ORIGINAL;
								forward_equ(*cnf, *ot, m_c, m_len, type);
							}
						}
					}
				}
				gid += stride_y();
			}
		}
		//======================================================//
		//                GPU Wrappers Definitions              //
		//======================================================//
		void initConstants(cuLimit culimit)
		{
			CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_limits), &culimit.limits, sizeof(uint32) * NLIMITS, 0, hipMemcpyHostToDevice));
		}
		void copyIf(uint32* dest, CNF* src, GSTATS* gstats)
		{
			if (profile_gpu) cutimer->start();
			reset_stats << <1, 1 >> > (gstats);
			uint32 nBlocks = std::min((inf.nClauses + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			copy_if_k << <nBlocks, BLOCK1D >> > (dest, src, gstats);
			if (profile_gpu) cutimer->stop(), cutimer->vo += cutimer->gpuTime();
			LOGERR("Copying literals failed");
			syncAll();
		}
		void calcScores(VARS* vars, uint32* hist)
		{
			if (profile_gpu) cutimer->start();
			uint32 nBlocks = std::min((inf.maxVar + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			assign_scores << <nBlocks, BLOCK1D >> > (vars->eligible, vars->scores, hist, inf.maxVar);
			if (profile_gpu) cutimer->stop(), cutimer->vo += cutimer->gpuTime();
			LOGERR("Assigning scores failed");
			syncAll();
		}
		void calcScores(VARS* vars, uint32* hist, OT* ot)
		{
			if (profile_gpu) cutimer->start();
			uint32 nBlocks = std::min((inf.maxVar + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			assign_scores << <nBlocks, BLOCK1D >> > (vars->eligible, vars->scores, hist, ot, inf.maxVar);
			if (profile_gpu) cutimer->stop(), cutimer->vo += cutimer->gpuTime();
			LOGERR("Assigning scores failed");
			syncAll();
		}
		void countMelted(VSTATE* vstate)
		{
			inf.n_del_vars_after = 0;
			forall_variables(v) {
				if (MELTED(vstate[v].state))
					inf.n_del_vars_after++;
			}
			assert(inf.n_del_vars_after >= inf.maxMelted);
			inf.n_del_vars_after -= inf.maxMelted;
			inf.maxMelted += inf.n_del_vars_after;
		}
		void countFinal(CNF* cnf, GSTATS* gstats, VSTATE* vstate)
		{
			reset_stats << <1, 1 >> > (gstats);
			const uint32 cnf_sz = inf.nClauses + (inf.nClauses >> 1);
			uint32 nBlocks = std::min((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUTereads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * (sizeof(uint32) + sizeof(uint32));
			cnt_cls_lits << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			countMelted(vstate);
			if (unified_access || sync_always) {
				LOGERR("Final CNF counting failed");
				syncAll();
			}
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
			inf.n_lits_after = hstats.numLits;
		}
		void countCls(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			const uint32 cnf_sz = inf.nClauses;
			uint32 nBlocks = std::min((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUTereads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * sizeof(uint32);
			cnt_cls << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
		}
		void countLits(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			const uint32 cnf_sz = inf.nClauses;
			uint32 nBlocks = std::min((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUTereads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * sizeof(uint32);
			cnt_lits << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_lits_after = hstats.numLits;
		}
		void countAll(CNF* cnf, GSTATS* gstats)
		{
			reset_stats << <1, 1 >> > (gstats);
			const uint32 cnf_sz = inf.nClauses + (inf.nClauses >> 1);
			uint32 nBlocks = std::min((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUTereads / (BLOCK1D << 1));
			uint32 smemSize = BLOCK1D * (sizeof(uint32) + sizeof(uint32));
			cnt_cls_lits << <nBlocks, BLOCK1D, smemSize >> > (cnf, gstats);
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost));
			inf.n_cls_after = hstats.numClauses;
			inf.n_lits_after = hstats.numLits;
		}
		void evalReds(CNF* cnf, GSTATS* gstats, VSTATE* vstate)
		{
			reset_stats << <1, 1 >> > (gstats);
			const uint32 cnf_sz = inf.nClauses + (inf.nClauses >> 1);
			uint32 nBlocks1 = std::min((cnf_sz + (BLOCK1D << 1) - 1) / (BLOCK1D << 1), maxGPUTereads / (BLOCK1D << 1));
			uint32 smemSize1 = BLOCK1D * sizeof(uint32) * 2;
			cnt_reds << <nBlocks1, BLOCK1D, smemSize1 >> > (cnf, gstats);
			countMelted(vstate);
			GSTATS hstats;
			CHECK(hipMemcpy(&hstats, gstats, sizeof(GSTATS), hipMemcpyDeviceToHost)); // avoids unified memory migration on large scale
			inf.n_cls_after = hstats.numClauses;
			inf.n_lits_after = hstats.numLits;
		}
		void cuMemSetAsync(addr_t mem, const Byte& val, const size_t& size)
		{
			uint32 nBlocks = std::min(uint32((size + BLOCK1D - 1) / BLOCK1D), maxGPUTereads / BLOCK1D);
			memset_k<Byte> << <nBlocks, BLOCK1D >> > (mem, val, size);
			if (sync_always) {
				LOGERR("CUDA memory set failed");
				syncAll();
			}
		}
		void prepareCNFAsync(CNF* cnf, const hipStream_t& _s)
		{
			assert(inf.nClauses);
			if (profile_gpu) cutimer->start(_s);
			uint32 nBlocks = std::min((inf.nClauses + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			prep_cnf_k << <nBlocks, BLOCK1D, 0, _s >> > (cnf);
			if (profile_gpu) cutimer->stop(_s), cutimer->sig += cutimer->gpuTime();
			if (sync_always) {
				LOGERR("Signature calculation failed");
				syncAll();
			}
		}
		void reduceOTAsync(CNF* cnf, OT* ot, const bool& p)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			if (profile_gpu) cutimer->start();
			uint32 nBlocks = std::min(uint32((inf.nDualVars + BLOCK1D - 1) / BLOCK1D), maxGPUTereads / BLOCK1D);
			reduce_ot << <nBlocks, BLOCK1D >> > (cnf, ot);
			if (p || sync_always) {
				LOGERR("Occurrence table reduction failed");
				syncAll();
				if (p) {
					PFLRULER('=', 30);
					PFLOG0("\toccurrence table");
					ot->print();
					PFLRULER('=', 30);
				}
			}
			if (profile_gpu) cutimer->stop(), cutimer->rot += cutimer->gpuTime();
		}
		void createOTAsync(CNF* cnf, OT* ot, const bool& p)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			if (profile_gpu) cutimer->start();
			uint32 rstGridSize = std::min(uint32((inf.nDualVars + BLOCK1D - 1) / BLOCK1D), maxGPUTereads / BLOCK1D);
			reset_ot_k << <rstGridSize, BLOCK1D >> > (ot);
			uint32 otGridSize = std::min((inf.nClauses + BLOCK1D - 1) / BLOCK1D, maxGPUTereads / BLOCK1D);
			create_ot_k << <otGridSize, BLOCK1D >> > (cnf, ot);
			if (p || sync_always) {
				LOGERR("Occurrence table creation failed");
				syncAll();
				assert(ot->accViolation());
				if (p) {
					PFLRULER('=', 30);
					PFLOG0("\toccurrence table");
					ot->print();
					PFLRULER('=', 30);
				}
			}
			if (profile_gpu) cutimer->stop(), cutimer->cot += cutimer->gpuTime();
		}
		void sortOTAsync(CNF* cnf, OT* ot, VARS* vars, hipStream_t* streams)
		{
			assert(cnf != NULL);
			assert(ot != NULL);
			assert(vars->numPVs);
			uint32 nBlocks = std::min((vars->numPVs + BLSORT - 1) / BLSORT, maxGPUTereads / BLSORT);
			hipStream_t s1, s2;
			if (profile_gpu) s1 = s2 = 0, cutimer->start();
			else s1 = streams[0], s2 = streams[1];
			sort_ot_p << <nBlocks, BLSORT, 0, s1 >> > (cnf, ot, vars->pVars);
			sort_ot_n << <nBlocks, BLSORT, 0, s2 >> > (cnf, ot, vars->pVars);
			if (sync_always) {
				LOGERR("Sorting OT failed");
				syncAll();
			}
			if (profile_gpu) cutimer->stop(), cutimer->sot += cutimer->gpuTime();
		}
		void veRollBack(CNF* cnf, OT* ot, VARS* vars, uint32* type)
		{
			// undo marked gate clauses if phase-2 in 've' ran out of memory which is an extreme case
			uint32 nBlocks = std::min((vars->numPVs + BLVE2 - 1) / BLVE2, maxGPUTereads / BLVE2);
			roll_back_gates << <nBlocks, BLVE2 >> > (cnf, ot, vars->pVars, type);
			LOGERR("BVE Roll-back failed");
			sync();
		}
		void veAsync(CNF* cnf, OT* ot, VARS* vars, hipStream_t* streams, cuMM& cumm, const cuHist& cuhist, const bool& in)
		{
			assert(vars->numPVs);
			if (profile_gpu) cutimer->start();
			if (!atomic_ve) {
				const uint32 cs_offset = cumm.pinnedCNF()->size();
				const S_REF data_offset = cumm.pinnedCNF()->data().size;
				S_REF* rref = cuhist.d_segs;
				uint32* type = cuhist.d_hist, *rpos = type + inf.maxVar;
				uint32* vorg = cuhist.d_vorg;
				// Phase-1
				uint32 smSize1 = (BLVE1 * SH_MAX_BVE_OUT1) * sizeof(uint32);
				#if VE_DBG
				if (in) in_ve_k_1 << <1, 1, smSize1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				else	   ve_k_1 << <1, 1, smSize1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				#else
				uint32 nBlocks1 = std::min((vars->numPVs + BLVE1 - 1) / BLVE1, maxGPUTereads / BLVE1);
				if (in) in_ve_k_1 << <nBlocks1, BLVE1, smSize1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				else	   ve_k_1 << <nBlocks1, BLVE1, smSize1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				#endif
				// Phase-2
				size_t tb1 = 0, tb2 = 0;
				void* ts1 = NULL, *ts2 = NULL;
				DeviceScan::ExclusiveScan(NULL, tb1, rpos, rpos, Sum(), cs_offset, vars->numPVs);
				DeviceScan::ExclusiveScan(NULL, tb2, rref, rref, Sum(), data_offset, vars->numPVs);
				LOGERR("BVE Phase-1 failed"); 
				sync(); //sync phase-1
				if (tb1 > cumm.literalsCap() || tb2 > cumm.scatterCap()) {
					addr_t tmpmem = cumm.allocTemp(tb1 + tb2);
					if (tmpmem == NULL) {
						veRollBack(cnf, ot, vars, type);
						return; 
					}
					ts1 = tmpmem, ts2 = tmpmem + tb1;
				}
				else ts1 = cuhist.d_lits, ts2 = cumm.scatter();
				DeviceScan::ExclusiveScan(ts1, tb1, rpos, rpos, Sum(), cs_offset, vars->numPVs, streams[0]);
				DeviceScan::ExclusiveScan(ts2, tb2, rref, rref, Sum(), data_offset, vars->numPVs, streams[1]);
				uint32 nBlocks2 = std::min((vars->numPVs + BLVE2 - 1) / BLVE2, maxGPUTereads / BLVE2);
				uint32 smSize2 = (BLVE2 * SH_MAX_BVE_OUT2) * sizeof(uint32);
				LOGERR("BVE Phase-2 failed"); 
				sync(streams[0]), sync(streams[1]); // sync phase-2
				#if VE_DBG
				ve_k_2 << <1, 1, smSize2 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				#else
				// Phase-3
				ve_k_2 << <nBlocks2, BLVE2, smSize2 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, vorg, type, rpos, rref);
				#endif
			}
			else {
				#if VE_DBG
				ve_k << <1, 1 >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, cuhist.d_vorg);
				#else
				uint32 nBlocks1 = std::min((vars->numPVs + BLVE - 1) / BLVE, maxGPUTereads / BLVE);
				ve_k << <nBlocks1, BLVE >> > (cnf, ot, vars->pVars, vars->units, vars->resolved, cuhist.d_vorg);
				#endif
			}
			if (profile_gpu) cutimer->stop(), cutimer->ve += cutimer->gpuTime();
			LOGERR("BVE Elimination failed"); 
			syncAll();
		}
		void hseAsync(CNF* cnf, OT* ot, VARS* vars)
		{
			assert(vars->numPVs);
			if (profile_gpu) cutimer->start();
#if SS_DBG
			putchar('\n');
			hse_k << <1, 1 >> > (cnf, ot, vars->pVars, vars->units);
#else
			uint32 nBlocks = std::min((vars->numPVs + BLHSE - 1) / BLHSE, maxGPUTereads / BLHSE);
			hse_k << <nBlocks, BLHSE >> > (cnf, ot, vars->pVars, vars->units);
#endif
			if (profile_gpu) cutimer->stop(), cutimer->hse += cutimer->gpuTime();
			if (sync_always) { 
				LOGERR("HSE Elimination failed");
				syncAll();
			}
		}
		void bceAsync(CNF* cnf, OT* ot, VARS* vars, const uint32* vorg)
		{
			assert(vars->numPVs);
			if (profile_gpu) cutimer->start();
			uint32 nBlocks = std::min((vars->numPVs + BLBCE - 1) / BLBCE, maxGPUTereads / BLBCE);
			bce_k << <nBlocks, BLBCE >> > (cnf, ot, vars->pVars, vars->resolved, vorg);
			if (profile_gpu) cutimer->stop(), cutimer->bce += cutimer->gpuTime();
			if (sync_always) {
				LOGERR("BCE Elimination failed");
				syncAll();
			}
		}
		void ereAsync(CNF* cnf, OT* ot, VARS* vars)
		{
			assert(vars->numPVs);
			if (profile_gpu) cutimer->start();
			dim3 block2D(devProp.warpSize, devProp.warpSize), grid2D(1, 1, 1);
			grid2D.y = std::min((vars->numPVs + block2D.y - 1) / block2D.y, maxGPUTereads / block2D.y);
			uint32 smemSize = devProp.warpSize * SH_MAX_ERE_OUT * sizeof(uint32);
			ere_k << <grid2D, block2D, smemSize >> > (cnf, ot, vars->pVars);
			if (profile_gpu) cutimer->stop(), cutimer->ere += cutimer->gpuTime();
			if (sync_always) {
				LOGERR("ERE Elimination failed");
				syncAll();
			}
		}

	}
}