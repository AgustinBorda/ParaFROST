#include "hip/hip_runtime.h"
/***********************************************************************[pfsimp.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "pfsimpopts.h"
#include "pfsolve.h"
#include "pfsort.h"

namespace pFROST {

	using namespace SIGmA;

	void ParaFROST::masterFree()
	{
		syncAll();
		cleanSigma();
		destroyStreams();
	}

	void ParaFROST::slavesFree()
	{

	}

	void ParaFROST::optSimp()
	{
		assert(sigma_en || sigma_live_en);
		ngpus = opt_gpus;
		nstreams = opt_streams;
		solve_en = opt_solve_en;
		ve_en = opt_ve_en || opt_ve_plus_en;
		ve_plus_en = opt_ve_plus_en;
		sub_en = opt_sub_en;
		bce_en = opt_bce_en;
		hre_en = opt_hre_en;
		all_en = opt_all_en;
		phases = opt_phases;
		mu_pos = opt_mu_pos;
		mu_neg = opt_mu_neg;
		lcve_min = opt_lcve_min;
		ve_round_min = opt_ve_round_min;
		ve_phase_min = opt_ve_phase_min;
		shrink_rate = opt_cnf_free;
		hse_limit = opt_hse_max_occurs;
		bce_limit = opt_bce_max_occurs;
		hre_limit = opt_hre_max_occurs;
		cls_en = all_en || sub_en || bce_en || hre_en;
		if (all_en) ve_en = 1, ve_plus_en = 1, sub_en = 1, bce_en = 1, hre_en = 1;
		if (!phases && ve_en) phases = 1; // at least 1 phase needed for BVE(+)
		if (phases && !ve_en) phases = 0;
		if (ngpus > devCount) ngpus = devCount;
	}

	void ParaFROST::extract(CNF* dest, WT& src)
	{
		for (uint32 lit = 2; lit < src.size(); lit++) {
			WL& ws = src.getClean(lit);
			if (ws.empty()) continue;
			for (WATCH* w = ws; w != ws.end(); w++) {
				CLAUSE& c = cm[w->ref];
				if (c.deleted()) continue;
				dest->newClause(c, sigma_live_en);
				inf.nClauses++, inf.nLiterals += c.size();
				c.markDeleted();
			}
		}
	}

	void ParaFROST::extract(CNF* dest, BCNF& src)
	{
		for (uint32 i = 0; i < src.size(); i++) {
			CLAUSE& c = cm[src[i]];
			if (c.deleted()) continue;
			dest->newClause(c, sigma_live_en);
			inf.nClauses++, inf.nLiterals += c.size();
		}
	}

	void ParaFROST::awaken()
	{
		// deal with any remained facts at root level
		PFLOG2(2, " Propagating any remaining facts before eliminations..");
		C_REF cref = BCP();
		assert(cref == NOREF); // dare to prove?!
		PFLOG2(2, " All good.");
		assert(DL() == ROOT_LEVEL);
		assert(conflict == NOREF);
		assert(cnfstate == UNSOLVED);
		assert(sp->propagated == trail.size());
		initSimp();
		if (sigma_live_en && trail.size() > sp->simplified) {
			PFLOGN2(2, " Shrinking CNF before eliminations..");
			shrinkWT();
			shrink(orgs);
			shrink(learnts);
			PFLENDING(2, 5, " (-%d variables)", trail.size() - sp->simplified);
			sp->simplified = trail.size();
		}
		if (orgs.empty()) { sigState = AWAKEN_FAIL; return; }
		// alloc simplifier memory 
		uint32 numCls = maxOrgs() + maxLearnts(), numLits = maxLiterals();
		if (phases) {
			inf.maxAddedCls = maxOrgs(), inf.maxAddedLits = maxOrgLits();
			PFLOG2(2, " Maximum added clauses/literals = %d/%d", inf.maxAddedCls, inf.maxAddedLits);
			numCls += inf.maxAddedCls, numLits += inf.maxAddedLits;
		}
		if (!cuMem.allocFixed(vars, numLits) ||
			!cuMem.resizeCNF(cnf, numCls, numLits)) 
		{ sigState = CNFALLOC_FAIL; return; }
		PFLOGN2(2, " Extracting clauses heterogeneously to device..");
		cuMem.resizeHostCNF(hcnf, maxOrgs() + maxLearnts(), maxLiterals());
		printStats(), inf.nClauses = inf.nLiterals = 0;
		extract(hcnf, wtBin), reflectCNF(streams[0], streams[1]), wtBin.clear(true);
		extract(hcnf, wt), reflectCNF(streams[0], streams[1]), wt.clear(true);
		extract(hcnf, orgs), reflectCNF(streams[0], streams[1]), orgs.clear(true);
		extract(hcnf, learnts), reflectCNF(streams[0], streams[1]), learnts.clear(true);
		// resize cnf & clean old database
		cuMem.resizeCNFAsync(cnf, hcnf);
		cm.destroy();
		sync(streams[0]), sync(streams[1]);
		assert(hcnf->data().size == off1);
		assert(inf.nClauses == hcnf->size() && hcnf->size() == off2);
		PFLENDING(2, 5, "(%d clauses extracted)", inf.nClauses);
		// compute clauses signatures
		sync(), calcSigCNFAsync(cnf, 0, inf.nClauses, streams[0]);
		// free host CNF
		cuMem.breakMirror();
		// compute histogram on GPU & alloc OT
		assert(inf.nDualVars);
		histogram.resize(inf.nDualVars);
		d_hist = thrust::raw_pointer_cast(histogram.data());
		h_hist = new uint32[inf.nDualVars];
		if (!reallocOT(streams[0])) return;
		cuMem.updateMaxCap(); // for monitoring GPU memory only
	}

	void ParaFROST::varReorder()
	{
		PFLOGN2(2, " Finding eligible variables for LCVE..");
		assert(d_hist != NULL);
		if (vars->nUnits) calcScores(vars, d_hist, ot); // update d_hist & calc scores
		else calcScores(vars, d_hist);
		CHECK(hipMemcpyAsync(h_hist, d_hist, inf.nDualVars * sizeof(uint32), hipMemcpyDeviceToHost, streams[2]));
		thrust::sort(thrust::cuda::par.on(streams[3]), vars->eligible, vars->eligible + inf.maxVar, GPU_LCV_CMP(vars->scores));
		sync(streams[2]);
		PFLDONE(2, 5);
		vars->nUnits = 0;
		if (verbose == 4) {
			PFLOG0(" Eligible variables:");
			CHECK(hipMemcpy(h_hist, d_hist, inf.nDualVars * sizeof(uint32), hipMemcpyDeviceToHost));
			for (uint32 v = 0; v < inf.maxVar; v++) {
				uint32 x = vars->eligible[v], p = v2l(x), n = neg(p);
				PFLOG1("  e[%d]->(v: %d, p: %d, n: %d, s: %d)", v, x, h_hist[p], h_hist[n], vars->scores[x]);
			}
		}
	}

	void ParaFROST::calcOccurs(const uint32& numLits)
	{
		PFLOGN2(2, " Copying remained literals..");
		assert(numLits);
		rawLits.resize(numLits);
		copyIf(thrust::raw_pointer_cast(rawLits.data()), cnf, vars->gstats);
		assert(vars->gstats->numLits == numLits);
		PFLENDING(2, 5, "(%d copied)", numLits);
		histSimp();
		rawLits.clear(), rawLits.shrink_to_fit();
	}

	void ParaFROST::histSimp()
	{
		PFLOGN2(2, " Computing histogram on %d literals..", rawLits.size());
		assert(rawLits.size());
		assert(histogram.size() == v2l(inf.maxVar + 1ULL));
		thrust::sort(rawLits.begin(), rawLits.end());
		thrust::counting_iterator<size_t> search_begin(0);
		thrust::upper_bound(rawLits.begin(), rawLits.end(), search_begin, search_begin + inf.nDualVars, histogram.begin());
		thrust::adjacent_difference(histogram.begin(), histogram.end(), histogram.begin());
		PFLDONE(2, 5);
	}

	void ParaFROST::preprocess()
	{
		timer.stop(), timer.solve += timer.cpuTime();
		if (!phases && !cls_en) return;
		/********************************/
		/*         awaken sigma         */
		/********************************/
		assert(cnfstate == UNSOLVED);
		assert(conflict == NOREF);
		timer.start();
		int phase;
		int64 lits_before, lits_diff;
		awaken();
		if (sigState == AWAKEN_FAIL ||
			sigState == CNFALLOC_FAIL) { timer.stop(); return; }
		if (sigState == OTALLOC_FAIL) goto writeBack;
		if (interrupted()) killSolver();
		/********************************/
		/*      1st-stage reduction     */
		/********************************/
		phase = 0, lits_before = inf.nLiterals, lits_diff = INT64_MAX;
		while (lits_diff > ve_phase_min && phase < phases) {
			if (interrupted()) killSolver();
			sync(streams[0]);
			if (!reallocCNF(phase + 1)) goto writeBack;
			createOTAsync(cnf, ot, 0);
			if (!prop()) killSolver();
			PFLOG2(2, "\t\tPhase-%d Variable Elections (p-mu: %d, n-mu: %d)",
				phase, mu_pos << vars->mu_inc, mu_neg << vars->mu_inc);
			if (!LCVE()) goto writeBack;
			VE(), cacheNumUnits(streams[3]);
			if (bce_en && phase) BCE();
			countAll();
			inf.nClauses = inf.n_cls_after, inf.nLiterals = inf.n_lits_after;
			lits_diff = lits_before - inf.nLiterals, lits_before = inf.nLiterals;
			cacheUnits(streams[3]);
			if (!reallocOT(streams[0])) goto writeBack;
			phase++, vars->mu_inc++;
		}
		sync(streams[0]);
		if (vars->nUnits || cls_en) createOTAsync(cnf, ot, 0);
		if (!prop()) killSolver();
		/********************************/
		/*      2nd-stage reduction     */
		/********************************/
		if (cls_en) {
			PFLOGN2(2," Initiating clause eliminations..");
			int t_p = mu_pos, t_n = mu_neg;
			while (t_p <= CE_POS_LMT && t_n <= CE_NEG_LMT) vars->mu_inc++, t_p <<= vars->mu_inc, t_n <<= vars->mu_inc;
			PFLDONE(2, 5);
			if (!LCVE()) goto writeBack;
			if (sub_en) SUB();
			if (bce_en) BCE();
			if (hre_en) HRE();
		}
		/********************************/
		/*           Write Back         */
		/********************************/
	writeBack:
		assert(sp->propagated == trail.size());
		if (interrupted()) killSolver();
		cacheCNF(streams[0], streams[1]);
		if (satisfied() || sigState == LCVE_FAIL || !inf.nClauses)
			cnfstate = SAT, printStats(1, 'p');
		else {
			if (canMap()) map(true);
			else assert(!mapped), newBeginning();
			if (sigma_live_en) {
				// update sigma trigger (inspired by Cadical)
				double w = weight(sigma_inc * (phase + 1));
				lrn.sigma_conf_max = nConflicts + w;
				PFLOG2(2, " SIGmA limit increased to %lld conflicts by a weight of %.2f", lrn.sigma_conf_max, w);
			}
		}
		stats.sigmifications++;
		if (inf.maxFrozen > sp->simplified) stats.n_forced += inf.maxFrozen - sp->simplified;
		assert(stats.n_forced <= inf.maxVar);
		timer.stop(), timer.simp += timer.cpuTime();
		if (!solve_en) killSolver();
		timer.start();
	}

	C_REF ParaFROST::newClause(SCLAUSE& s)
	{
		assert(!s.deleted());
		C_REF r = s.ref();
		if (r == NOREF) {
			int sz = s.size();
			assert(sz > 1);
			r = cm.alloc(sz);
			s.set_ref(r); // new ref overwrites simplified clause sig.
			CLAUSE& new_c = cm[r];
			if (mapped) vmap.mapClause(new_c, s);
			else new_c.copyLitsFrom(s);
			assert(sz == new_c.size());
			assert(new_c[0] > 0 && new_c[1] > 0);
			assert(new_c[0] <= UINT32_MAX && new_c[1] <= UINT32_MAX);
			new_c.set_status(s.status());
			if (sz == 2) {
				if (s.original()) inf.nOrgBins++;
				else assert(s.learnt()), inf.nLearntBins++;
			}
			else {
				assert(sz > 2);
				if (s.learnt()) {
					new_c.set_LBD(new_c.size());
					learnts.push(r);
					inf.nLearntLits += sz;
				}
				else {
					orgs.push(r);
					inf.nLiterals += sz;
				}
			}
		}
		return r;
	}

	void ParaFROST::newBeginning() {
		assert(sigma_en || sigma_live_en);
		assert(!hcnf->empty());
		assert(wtBin.empty()), assert(wt.empty());
		assert(orgs.empty()), assert(learnts.empty());
		inf.nOrgBins = inf.nLearntBins = 0;
		inf.nLiterals = inf.nLearntLits = 0;
		assert(inf.maxVar > vmap.numVars());
		uint32 tableSize = mapped ? v2l(vmap.size()) : inf.nDualVars;
		wtBin.resize(tableSize), wt.resize(tableSize);
		cm.init(hcnf->data().size);
		if (!mapped) assert(vmap.empty()), sp->lockMelted(inf.maxVar);
		sync(streams[0]), sync(streams[1]); // sync CNF caching
		cacheResolved(streams[2]), createWT(), copyWatched(), copyNonWatched();  // must follow this order
		syncAll();
		inf.nOrgCls = inf.nClauses = orgs.size();
		inf.nOrgLits = inf.nLiterals;
		printStats(1, 'p');
	}

	bool ParaFROST::LCVE()
	{
		// reorder variables
		varReorder();
		// extended LCVE
		PFLOGN2(2, " Electing variables..");
		vars->numPVs = 0, vars->pVars->clear();
		for (uint32 i = 0; i < inf.maxVar; i++) {
			uint32 cand = vars->eligible[i];
			assert(cand && cand <= inf.maxVar);
			if (sp->vstate[cand] == FROZEN || sp->vstate[cand] == MELTED) continue;
			if (sp->frozen[cand]) continue;
			uint32 p = v2l(cand), n = neg(p);
			assert((*ot)[p].size() == h_hist[p]);
			assert((*ot)[n].size() == h_hist[n]);
			if (h_hist[p] == 0 && h_hist[n] == 0) continue;
			uint32 pos_temp = mu_pos << vars->mu_inc, neg_temp = mu_neg << vars->mu_inc;
			if (h_hist[p] >= pos_temp && h_hist[n] >= neg_temp) break;
			assert(sp->vstate[cand] == ACTIVE);
			vars->pVars->_push(cand);
			depFreeze((*ot)[p], cand, pos_temp, neg_temp);
			depFreeze((*ot)[n], cand, pos_temp, neg_temp);
		}
		vars->numPVs = vars->pVars->size();
		assert(verifyLCVE());
		memset(sp->frozen, 0, inf.maxVar + 1ULL);
		if (vars->numPVs) {
			uint32 mcv = vars->pVars->back();
			PFLENDING(2, 5, "(%d elected, mcv: %d, pH: %d, nH: %d)", vars->numPVs, mcv, h_hist[v2l(mcv)], h_hist[neg(v2l(mcv))]);
			if (verbose == 4) { PFLOGN0(" PLCVs "); printVars(*vars->pVars, vars->numPVs, 'v'); }
		}
		else {
			PFLDONE(2, 5);
			// NOTE: practically and perfhaps theoretically if LCVE couldn't elect
			// any variable, that means all variables are eliminated, propagated, or
			// more interestingly disappeared in clause eliminations which is enough
			// to prove the formula is SATISFIABLE
			sigState = LCVE_FAIL;
		}
		if (vars->numPVs < lcve_min) {
			if (verbose > 1) PFLOGW("parallel variables not enough -> skip SIGmA");
			return false;
		}
		return true;
	}

	bool ParaFROST::propClause(SCLAUSE& c, const uint32& f_assign)
	{
		uint32 sig = 0;
		int n = 0;
		bool check = false;
		for (int k = 0; k < c.size(); k++) {
			uint32 lit = c[k];
			if (lit != f_assign) {
				if (isTrue(lit)) return true;
				c[n++] = lit;
				sig |= MAPHASH(lit);
			}
			else check = true;
		}
		assert(check);
		assert(n == c.size() - 1);
		assert(c.hasZero() < 0);
		assert(c.isSorted());
		c.set_sig(sig);
		c.pop();
		return false;
	}

	bool ParaFROST::prop()
	{
		if (!enqeueCached(streams[3])) { cnfstate = UNSAT; return false; }
		while (sp->propagated < trail.size()) { // propagate units
			uint32 assign = trail[sp->propagated++], f_assign = flip(assign);
			assert(assign);
			PFLBCP(this, 4, assign);
			OL& ol = (*ot)[assign], & f_ol = (*ot)[f_assign];
			for (uint32 i = 0; i < ol.size(); i++) (*cnf)[ol[i]].markDeleted(); // remove satisfied
			for (uint32 i = 0; i < f_ol.size(); i++) { // reduce unsatisfied 
				SCLAUSE& c = (*cnf)[f_ol[i]];
				assert(c.size());
				if (c.deleted() || propClause(c, f_assign)) continue; // clause satisfied
				assert(c.size()); // cannot be empty at this point
				if (c.size() == 1) {
					assert(*c > 1);
					if (unassigned(*c)) enqueue(*c); 
					else { cnfstate = UNSAT; return false; }  // conflict on top level
				}
			}
			(*ot)[assign].clear(true), (*ot)[f_assign].clear(true);
		}
		cleanProped();
		return true;
	}

	void ParaFROST::VE()
	{
		if (interrupted()) killSolver();
		int64 lits_before = inf.nLiterals, lits_removed = INT64_MAX;
		int round = 0;
		while (lits_removed > ve_round_min) {
			PFLOG2(2, " Elimination round %d:", round);
			if (ve_plus_en) {
				PFLOGN2(2, "  1) HSE-ing variables..");
				hse(cnf, ot, vars, hse_limit), countLits();
				lits_removed = lits_before - inf.n_lits_after;
				assert(lits_removed >= 0);
				PFLENDING(2, 5, "(Literals removed : %lld)", -lits_removed);
				if (round && !lits_removed) break;
				lits_before = inf.n_lits_after;
			}
			PFLOGN2(2, "  2) Eliminating variables..");
			ve(cnf, ot, vars, sigma_live_en), countLits();
			lits_removed = lits_before - inf.n_lits_after;
			PFLENDING(2, 5, "(Literals removed : %c%lld)", lits_removed < 0 ? '+' : '-', abs(lits_removed));
			lits_before = inf.n_lits_after;
			if (filterPVs()) break;
			round++;
		}
		PFLREDALL(this, 2, "BVE(+) Reductions");
	}

	void ParaFROST::SUB()
	{
		if (interrupted()) killSolver();
		PFLOGN2(2, " SUB-ing variables..");
		hse(cnf, ot, vars, hse_limit);
		cacheNumUnits(streams[3]);
		cacheUnits(streams[3]);
		PFLDONE(2, 5);
		PFLREDCL(this, 2, "SUB Reductions");
		if (!prop()) killSolver();
	}

	void ParaFROST::BCE()
	{
		if (interrupted()) killSolver();
		PFLOGN2(2, " Eliminating blocked clauses..");
		bce(cnf, ot, vars, bce_limit);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "BCE Reductions");
	}

	void ParaFROST::HRE()
	{
		if (interrupted()) killSolver();
		PFLOGN2(2, " Eliminating hidden redundances..");
		hre(cnf, ot, vars, hre_limit);
		PFLDONE(2, 5);
		PFLREDCL(this, 2, "HRE Reductions");
	}

	inline void	ParaFROST::initSimp() {
		nForced = 0, sigState = AWAKEN_SUCC;
		off1 = off2 = 0;
		// free old memory (keeps the streams)
		if (cuMem.capacity()) cleanSigma();
	}

	inline bool ParaFROST::enqeueCached(const hipStream_t& stream) {
		if (vars->nUnits) {
			nForced = sp->propagated;
			sync(stream); // sync units copy
			assert(vars->cachedUnits != NULL);
			uint32* t = vars->cachedUnits + vars->nUnits;
			for (uint32* u = vars->cachedUnits; u != t; u++) {
				LIT_ST val = value(*u);
				if (val == UNDEFINED) enqueue(*u);
				else if (!val) return false; // early conflict detection
			}
			if (trail.size() == sp->propagated) vars->nUnits = nForced = 0; // duplicate units
			else PFLTRAIL(this, 3);
			sync(); // sync ot creation
		}
		return true;
	}

	inline void ParaFROST::cacheCNF(const hipStream_t& s1, const hipStream_t& s2)
	{
		// sync any gpu streams still running
		syncAll(); 
		// 0) count clauses on GPU and variables on CPU
		countFinal();
		if (!inf.nClauses) return; // all eliminated
		cuMem.mirrorCNF(hcnf);
		// 1) copy actual cnf data async.
		CHECK(hipMemcpyAsync(hcnf->data().mem, cuMem.cnfDatadPtr(), hcnf->data().size * sizeof(S_REF), hipMemcpyDeviceToHost, s1));
		// 2) sort cs w.r.t  clause status on gpu
		thrust::stable_sort(thrust::cuda::par.on(s2), cuMem.cnfClsdPtr(), cuMem.cnfClsdPtr() + hcnf->size(), CNF_CMP_ST(cnf));
		// 3) sort cs w.r.t clause size on gpu (inf.nClauses gives nr. undeleted clauses)
		hcnf->resize(inf.nClauses); // must be done after step (2)
		thrust::stable_sort(thrust::cuda::par.on(s2), cuMem.cnfClsdPtr(), cuMem.cnfClsdPtr() + inf.nClauses, CNF_CMP_SZ(cnf));
		// 4) copy sorted cs async.
		CHECK(hipMemcpyAsync(hcnf->csData(), cuMem.cnfClsdPtr(), inf.nClauses * sizeof(S_REF), hipMemcpyDeviceToHost, s2));
	}

	inline void ParaFROST::depFreeze(const OL& ol, const uint32& cand, const uint32& p_temp, const uint32& n_temp)
	{
		for (uint32 i = 0; i < ol.size(); i++) {
			SCLAUSE& c = (*cnf)[ol[i]];
			for (int k = 0; k < c.size(); k++) {
				register uint32 v = l2a(c[k]), p = v2l(v), n = neg(p);
				if (v != cand && (h_hist[p] < p_temp || h_hist[n] < n_temp)) sp->frozen[v] = 1;
			}
		}
	}

	inline void	ParaFROST::cleanProped() {
		if (vars->nUnits) {
			nForced = sp->propagated - nForced;
			PFLREDALL(this, 2, "BCP Reductions");
			nForced = 0, vars->tmpObj.clear();
			assert(vars->tmpObj.data() == cuMem.unitsdPtr());
			CHECK(hipMemcpyAsync(vars->units, &vars->tmpObj, sizeof(cuVecU), hipMemcpyHostToDevice));
			reduceOTAsync(cnf, ot, 0);
		}
		else sync(); // sync ot creation
	}

	inline void	ParaFROST::cleanSigma() {
		histogram.clear(), histogram.shrink_to_fit();
		if (h_hist != NULL) delete[] h_hist;
		if (vars != NULL) delete vars;
		vars = NULL, h_hist = NULL, d_hist = NULL;
		cuMem.destroy(), ot = NULL, cnf = NULL, hcnf = NULL;
	}

}