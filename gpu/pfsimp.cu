/***********************************************************************[pfsimp.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "pfsimpopts.h"
#include "pfsolve.h"
#include "pfsort.h"
#include <cub/device/device_select.cuh>
using namespace hipcub;

namespace pFROST {
	using namespace SIGmA;
	cuTIMER *cutimer = NULL;
	bool unified_access = false;
	bool profile_gpu = false;
	bool sync_always = false;
	bool atomic_ve = false;
	bool gc_par = false;

	void ParaFROST::masterFree()
	{
		syncAll();
		cleanFixed();
		cleanDynamic();
		destroyStreams();
		cumem.breakMirror(), hcnf = NULL;
	}

	void ParaFROST::slavesFree()
	{

	}

	void ParaFROST::optSimp()
	{
		assert(sigma_en || sigma_live_en);
		ngpus = opt_gpus;
		sub_en = opt_sub_en;
		bce_en = opt_bce_en;
		hre_en = opt_hre_en;
		all_en = opt_all_en;
		phases = opt_phases;
		mu_pos = opt_mu_pos;
		mu_neg = opt_mu_neg;
		gc_par = opt_gc_par_en;
		nstreams = opt_streams;
		solve_en = opt_solve_en;
		lcve_min = opt_lcve_min;
		shrink_rate = opt_cnf_free;
		ve_plus_en = opt_ve_plus_en;
		atomic_ve = opt_atomic_ve_en;
		sort_cnf_en = opt_sort_cnf_en;
		xor_limit = opt_xor_max_arity;
		hse_limit = opt_hse_max_occurs;
		bce_limit = opt_bce_max_occurs;
		hre_limit = opt_hre_max_occurs;
		ve_phase_min = opt_ve_phase_min;
		profile_gpu = opt_profile_gpu_en;
		sync_always = opt_sync_always_en;
		ve_en = opt_ve_en || opt_ve_plus_en;
		unified_access = opt_unified_access_en;
		cls_en = all_en || sub_en || bce_en || hre_en;
		if (all_en) ve_en = 1, ve_plus_en = 1, sub_en = 1, bce_en = 1, hre_en = 1;
		if (!phases && ve_en) phases = 1; // at least 1 phase needed for BVE(+)
		if (phases && !ve_en) phases = 0;
		if (ngpus > devCount) ngpus = devCount;
		if (profile_gpu) cutimer = new cuTIMER;
	}

	void ParaFROST::extract(CNF* dest, WT& src)
	{
		for (uint32 lit = 2; lit < src.size(); lit++) {
			WL& ws = src.getClean(lit);
			if (ws.empty()) continue;
			for (WATCH* w = ws; w != ws.end(); w++) {
				CLAUSE& c = cm[w->ref];
				if (c.deleted()) continue;
				dest->newClause(c, sigma_live_en);
				inf.nClauses++, inf.nLiterals += c.size();
				c.markDeleted();
			}
		}
	}

	void ParaFROST::extract(CNF* dest, BCNF& src)
	{
		for (uint32 i = 0; i < src.size(); i++) {
			CLAUSE& c = cm[src[i]];
			if (c.deleted()) continue;
			dest->newClause(c, sigma_live_en);
			inf.nClauses++, inf.nLiterals += c.size();
		}
	}

	void ParaFROST::awaken()
	{
		// deal with any remained facts at root level
		PFLOG2(2, " Propagating any remaining facts before eliminations..");
		C_REF cref = BCP();
		assert(cref == NOREF); // dare to prove?!
		PFLOG2(2, " All good.");
		assert(DL() == ROOT_LEVEL);
		assert(conflict == NOREF);
		assert(cnfstate == UNSOLVED);
		assert(sp->propagated == trail.size());
		initSimp();
		if (sigma_live_en && trail.size() > sp->simplified) shrink();
		if (orgs.empty()) { sigState = AWAKEN_FAIL; return; }
		// alloc simplifier memory 
		uint32 numCls = maxOrgs() + maxLearnts(), numLits = maxLiterals();
		if (phases) {
			inf.maxAddedCls = maxOrgs(), inf.maxAddedLits = maxOrgLits();
			PFLOG2(2, " Maximum added clauses/literals = %d/%d", inf.maxAddedCls, inf.maxAddedLits);
			numCls += inf.maxAddedCls, numLits += inf.maxAddedLits;
		}
		assert(inf.nDualVars);
		if (!cumem.resizeCNF(cnf, numCls, numLits) ||
			!cumem.allocHist(cuhist, numLits) ||
			!cumem.allocVars(vars, numLits) ||
			!cumem.allocAux(numCls)) 
		{ sigState = CNFALLOC_FAIL; return; }
		if (unified_access) {
			PFLOGN2(2, " Extracting clauses directly to device..");
			if (profile_gpu) cutimer->start();
			printStats(), inf.nClauses = inf.nLiterals = 0;
			extract(cnf, wtBin), wtBin.clear(true);
			extract(cnf, wt), wt.clear(true);
			extract(cnf, orgs), orgs.clear(true);
			extract(cnf, learnts), learnts.clear(true);
			cm.destroy();
			assert(inf.nClauses == cnf->size());
			if (profile_gpu) cutimer->stop(), cutimer->io += cutimer->gpuTime();
		}
		else {
			PFLOGN2(2, " Extracting clauses heterogeneously to device..");
			cumem.createMirror(hcnf, maxOrgs() + maxLearnts(), maxLiterals());
			if (profile_gpu) cutimer->start(streams[0]);
			printStats(), inf.nClauses = inf.nLiterals = 0;
			extract(hcnf, wtBin), reflectCNF(streams[0], streams[1]), wtBin.clear(true);
			extract(hcnf, wt), reflectCNF(streams[0], streams[1]), wt.clear(true);
			extract(hcnf, orgs), reflectCNF(streams[0], streams[1]), orgs.clear(true);
			extract(hcnf, learnts), reflectCNF(streams[0], streams[1]), learnts.clear(true);
			// resize cnf & clean old database
			cumem.resizeCNFAsync(cnf, hcnf);
			cm.destroy();
			sync(streams[0]), sync(streams[1]);
			assert(hcnf->data().size == off1);
			assert(inf.nClauses == hcnf->size() && hcnf->size() == off2);
			if (profile_gpu) cutimer->stop(streams[0]), cutimer->io += cutimer->gpuTime();
		}
		PFLENDING(2, 5, "(%d clauses extracted)", inf.nClauses);
		// compute clauses signatures
		sync(), calcSigCNFAsync(cnf, 0, inf.nClauses, streams[0]);
		// compute histogram on GPU & alloc OT
		if (!reallocOT(streams[0])) return;
		cumem.updateMaxCap(); // for reporting GPU memory only
	}

	void ParaFROST::varReorder()
	{
		PFLOGN2(2, " Finding eligible variables for LCVE..");
		assert(cuhist.d_hist != NULL);
		// NOTE: OT creation will be synced in calcScores call
		if (vars->nUnits) calcScores(vars, cuhist.d_hist, ot); // update d_hist & calc scores
		else calcScores(vars, cuhist.d_hist);
		cuhist.cacheHist(streams[2]);
		if (profile_gpu) cutimer->start(streams[3]);
		thrust::sort(thrust::cuda::par.on(streams[3]), vars->eligible, vars->eligible + inf.maxVar, GPU_LCV_CMP(vars->scores));
		PFLDONE(2, 5);
		vars->nUnits = 0;
		sync(streams[2]);
		if (profile_gpu) cutimer->stop(streams[3]), cutimer->vo += cutimer->gpuTime();
		if (verbose == 4) {
			PFLOG0(" Eligible variables:");
			for (uint32 v = 0; v < inf.maxVar; v++) {
				uint32 x = vars->eligible[v], p = v2l(x), n = neg(p);
				PFLOG1("  e[%d]->(v: %d, p: %d, n: %d, s: %d)", v, x, cuhist[p], cuhist[n], vars->scores[x]);
			}
		}
	}

	void ParaFROST::calcOccurs(const uint32& numLits)
	{
		assert(numLits);
		PFLOGN2(2, " Copying remained literals..");
		copyIf(cuhist.d_lits, cnf, vars->gstats);
		assert(vars->gstats->numLits == numLits);
		PFLENDING(2, 5, "(%d copied)", numLits);
		histSimp(numLits);
	}

	void ParaFROST::histSimp(const uint32& numLits)
	{
		assert(numLits);
		PFLOGN2(2, " Computing histogram on %d elements..", numLits);
		if (profile_gpu) cutimer->start();
		thrust::sort(cuhist.thrust_lits, cuhist.thrust_lits + numLits);
		thrust::counting_iterator<size_t> search_begin(0);
		thrust::upper_bound(cuhist.thrust_lits, cuhist.thrust_lits + numLits, search_begin, search_begin + inf.nDualVars, cuhist.thrust_hist);
		thrust::adjacent_difference(cuhist.thrust_hist, cuhist.thrust_hist + inf.nDualVars, cuhist.thrust_hist);
		if (profile_gpu) cutimer->stop(), cutimer->vo += cutimer->gpuTime();
		PFLDONE(2, 5);
	}

	void ParaFROST::sigmify()
	{
		timer.stop(), timer.solve += timer.cpuTime();
		if (!phases && !cls_en) return;
		/********************************/
		/*         awaken sigma         */
		/********************************/
		assert(cnfstate == UNSOLVED);
		assert(conflict == NOREF);
		timer.start();
		int phase;
		int64 lits_before, lits_diff;
		awaken();
		if (sigState == AWAKEN_FAIL ||
			sigState == CNFALLOC_FAIL) { timer.stop(); return; }
		if (sigState == OTALLOC_FAIL) goto writeBack;
		if (interrupted()) killSolver();
		/********************************/
		/*      1st-stage reduction     */
		/********************************/
		phase = 0, lits_before = inf.nLiterals, lits_diff = INT64_MAX;
		while ((lits_diff > ve_phase_min || phase < 2) && phase < phases) {
			if (interrupted()) killSolver();
			sync(streams[0]);
			if (!reallocCNF(phase + 1)) goto writeBack;
			createOTAsync(cnf, ot, 0);
			if (!prop()) killSolver();
			PFLOG2(2, "\t\tPhase-%d Variable Elections (p-mu: %d, n-mu: %d)",
				phase, mu_pos << vars->mu_inc, mu_neg << vars->mu_inc);
			if (!LCVE()) goto writeBack;
			//(*vars->pVars)[0] = 3, (*vars->pVars)[1] = 5, (*vars->pVars)[2] = 13, vars->numPVs = 3, vars->pVars->resize(vars->numPVs);
			sortOTAsync(cnf, ot, vars, streams);
			if (ve_en) VE();
			if (bce_en) BCE();
			countAll(), filterPVs();
			cacheNumUnits(streams[3]);
			inf.nClauses = inf.n_cls_after, inf.nLiterals = inf.n_lits_after;
			lits_diff = lits_before - inf.nLiterals, lits_before = inf.nLiterals;
			cacheUnits(streams[3]);
			if (!reallocOT(streams[0])) goto writeBack;
			phase++, vars->mu_inc++;
		}
		sync(streams[0]);
		if (vars->nUnits || cls_en) createOTAsync(cnf, ot, 0);
		if (!prop()) killSolver();
		/********************************/
		/*      2nd-stage reduction     */
		/********************************/
		if (cls_en) {
			PFLOGN2(2," Initiating clause eliminations..");
			int t_p = mu_pos, t_n = mu_neg;
			while (t_p <= CE_POS_LMT && t_n <= CE_NEG_LMT) vars->mu_inc++, t_p <<= vars->mu_inc, t_n <<= vars->mu_inc;
			PFLDONE(2, 5);
			if (!LCVE()) goto writeBack;
			sortOTAsync(cnf, ot, vars, streams);
			if (sub_en) SUB();
			if (hre_en) HRE();
		}
		/********************************/
		/*           Write Back         */
		/********************************/
	writeBack:
		assert(sp->propagated == trail.size());
		if (interrupted()) killSolver();
		cacheCNF(streams[0], streams[1]);
		if (inf.maxFrozen > sp->simplified) stats.n_forced += inf.maxFrozen - sp->simplified;
		assert(stats.n_forced <= inf.maxVar);
		stats.sigmifications++;
		if (satisfied() || !inf.nClauses)
			cnfstate = SAT, printStats(1, 'p');
		else {
			if (canMap()) map(true);
			else assert(!mapped), newBeginning();
			if (sigma_live_en) {
				// update sigma trigger (inspired by Cadical) 
				// but we decrease phases and reset last shrinked too
				double current_inc = sigma_inc * (phase + 1);
				sigmaWeight(current_inc);
				lrn.lastsimplified = 0;
				lrn.sigma_conf_max = nConflicts + current_inc;
				PFLOG2(2, " SIGmA limit increased to %lld conflicts by a weight of %.2f", lrn.sigma_conf_max, current_inc);
				if (phases > 1) {
					phases--;
					PFLOG2(2, " SIGmA phases decreased to %d phases", phases);
				}
			}
		}
		timer.stop(), timer.simp += timer.cpuTime();
		if (!solve_en) killSolver();
		timer.start();
	}

	C_REF ParaFROST::newClause(SCLAUSE& s)
	{
		assert(!s.deleted());
		C_REF r = s.ref();
		if (r == NOREF) {
			int sz = s.size();
			assert(sz > 1);
			r = cm.alloc(sz);
			s.set_ref(r); // new ref overwrites simplified clause sig.
			CLAUSE& new_c = cm[r];
			if (mapped) vmap.mapClause(new_c, s);
			else new_c.copyLitsFrom(s);
			assert(sz == new_c.size());
			assert(new_c[0] > 0 && new_c[1] > 0);
			assert(new_c[0] <= UINT32_MAX && new_c[1] <= UINT32_MAX);
			new_c.set_status(s.status());
			if (sz == 2) {
				if (s.original()) inf.nOrgBins++;
				else assert(s.learnt()), inf.nLearntBins++;
			}
			else {
				assert(sz > 2);
				if (s.learnt()) {
					new_c.set_lbd(sz);
					learnts.push(r);
					inf.nLearntLits += sz;
				}
				else {
					orgs.push(r);
					inf.nLiterals += sz;
				}
			}
		}
		return r;
	}

	void ParaFROST::newBeginning() {
		assert(sigma_en || sigma_live_en);
		assert(wtBin.empty()), assert(wt.empty());
		assert(orgs.empty()), assert(learnts.empty());
		inf.nOrgBins = inf.nLearntBins = 0;
		inf.nLiterals = inf.nLearntLits = 0;
		assert(inf.maxVar > vmap.numVars());
		uint32 tableSize = mapped ? v2l(vmap.size()) : inf.nDualVars;
		wtBin.resize(tableSize), wt.resize(tableSize);
		if (!mapped) assert(vmap.empty()), sp->lockMelted(inf.maxVar);
		if (unified_access) {
			assert(hcnf == NULL); // assert we don't have memory leak
			hcnf = cnf;
			cm.init(hcnf->data().size * 1.5);
			cacheResolved(streams[2]);
		}
		else {
			assert(!hcnf->empty());
			cm.init(hcnf->data().size * 1.5);
			sync(streams[0]), sync(streams[1]); // sync CNF caching
			cacheResolved(streams[2]);
			if (profile_gpu) cutimer->stop(streams[1]), cutimer->io += cutimer->gpuTime();
		}
		createWT(), copyWatched(), copyNonWatched();  // must follow this order
		syncAll();
		inf.nOrgCls = inf.nClauses = orgs.size();
		inf.nOrgLits = inf.nLiterals;
		printStats(1, 'p');
		if (unified_access) {
			hcnf = NULL;
			if (profile_gpu) cutimer->stop(), cutimer->io += cutimer->gpuTime();
		}
	}

	bool ParaFROST::LCVE()
	{
		// reorder variables
		varReorder();
		// extended LCVE
		PFLOGN2(2, " Electing variables..");
		vars->numPVs = 0, vars->pVars->clear();
		for (uint32 i = 0; i < inf.maxVar; i++) {
			uint32 cand = vars->eligible[i];
			assert(cand && cand <= inf.maxVar);
			if (sp->vstate[cand] == FROZEN || sp->vstate[cand] == MELTED) continue;
			if (sp->frozen[cand]) continue;
			uint32 p = v2l(cand), n = neg(p);
			assert((*ot)[p].size() == cuhist[p]);
			assert((*ot)[n].size() == cuhist[n]);
			if (cuhist[p] == 0 && cuhist[n] == 0) continue;
			uint32 pos_temp = mu_pos << vars->mu_inc, neg_temp = mu_neg << vars->mu_inc;
			if (cuhist[p] >= pos_temp && cuhist[n] >= neg_temp) break;
			assert(sp->vstate[cand] == ACTIVE);
			vars->pVars->_push(cand);
			depFreeze((*ot)[p], cand, pos_temp, neg_temp);
			depFreeze((*ot)[n], cand, pos_temp, neg_temp);
		}
		vars->numPVs = vars->pVars->size();
		assert(verifyLCVE());
		memset(sp->frozen, 0, inf.maxVar + 1ULL);
		if (vars->numPVs) {
			uint32 mcv = vars->pVars->back();
			PFLENDING(2, 5, "(%d elected, mcv: %d, pH: %d, nH: %d)", vars->numPVs, mcv, cuhist[v2l(mcv)], cuhist[neg(v2l(mcv))]);
			if (verbose == 4) { PFLOGN0(" PLCVs "); printVars(*vars->pVars, vars->numPVs, 'v'); }
		}
		if (vars->numPVs < lcve_min) {
			PFLDONE(2, 5);
			if (verbose > 1) PFLOGW("parallel variables not enough -> skip SIGmA");
			return false;
		}
		return true;
	}

	bool ParaFROST::propClause(SCLAUSE& c, const uint32& f_assign)
	{
		uint32 sig = 0;
		int n = 0;
		bool check = false;
		for (int k = 0; k < c.size(); k++) {
			uint32 lit = c[k];
			if (lit != f_assign) {
				if (isTrue(lit)) return true;
				c[n++] = lit;
				sig |= MAPHASH(lit);
			}
			else check = true;
		}
		assert(check);
		assert(n == c.size() - 1);
		assert(c.hasZero() < 0);
		assert(c.isSorted());
		c.set_sig(sig);
		c.pop();
		return false;
	}

	bool ParaFROST::prop()
	{
		if (!enqeueCached(streams[3])) { cnfstate = UNSAT; return false; }
		while (sp->propagated < trail.size()) { // propagate units
			uint32 assign = trail[sp->propagated++], f_assign = flip(assign);
			assert(assign);
			PFLBCP(this, 4, assign);
			OL& ol = (*ot)[assign], & f_ol = (*ot)[f_assign];
			for (uint32 i = 0; i < ol.size(); i++) (*cnf)[ol[i]].markDeleted(); // remove satisfied
			for (uint32 i = 0; i < f_ol.size(); i++) { // reduce unsatisfied 
				SCLAUSE& c = (*cnf)[f_ol[i]];
				assert(c.size());
				if (c.deleted() || propClause(c, f_assign)) continue; // clause satisfied
				assert(c.size()); // cannot be empty at this point
				if (c.size() == 1) {
					assert(*c > 1);
					if (unassigned(*c)) enqueue(*c); 
					else { cnfstate = UNSAT; return false; }  // conflict on top level
				}
			}
			(*ot)[assign].clear(true), (*ot)[f_assign].clear(true);
		}
		cleanProped();
		return true;
	}

	void ParaFROST::VE()
	{
		if (interrupted()) killSolver();
		uint32 cs_size = cumem.pinnedCNF()->size(), data_size = cumem.pinnedCNF()->data().size;
		if (ve_plus_en) {
			PFLOGN2(2, "  Eliminating (self)-subsumptions..");
			hseAsync(cnf, ot, vars, hse_limit);
			PFLDONE(2, 5);
		}
		PFLOGN2(2, "  Eliminating variables..");
		veAsync(cnf, ot, vars, streams, cuhist, cs_size, data_size, xor_limit, sigma_live_en);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "BVE(+) Reductions");
	}

	void ParaFROST::SUB()
	{
		if (interrupted()) killSolver();
		if (!vars->numPVs) return;
		PFLOGN2(2, " SUB-ing variables..");
		hseAsync(cnf, ot, vars, hse_limit);
		cacheNumUnits(streams[3]);
		cacheUnits(streams[3]);
		PFLDONE(2, 5);
		PFLREDCL(this, 2, "SUB Reductions");
		if (!prop()) killSolver();
	}

	void ParaFROST::BCE()
	{
		if (interrupted()) killSolver();
		if (!vars->numPVs) return;
		PFLOGN2(2, " Eliminating blocked clauses..");
		bceAsync(cnf, ot, vars, bce_limit);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "BCE Reductions");
	}

	void ParaFROST::HRE()
	{
		if (interrupted()) killSolver();
		if (!vars->numPVs) return;
		PFLOGN2(2, " Eliminating hidden redundances..");
		hreAsync(cnf, ot, vars, hre_limit);
		PFLDONE(2, 5);
		PFLREDCL(this, 2, "HRE Reductions");
	}

	inline void	ParaFROST::initSimp() {
		nForced = 0, sigState = AWAKEN_SUCC;
		off1 = off2 = 0;
		if (cumem.ucapacity()) cleanDynamic();
	}

	inline void ParaFROST::depFreeze(const OL& ol, const uint32& cand, const uint32& p_temp, const uint32& n_temp)
	{
		for (uint32 i = 0; i < ol.size(); i++) {
			SCLAUSE& c = (*cnf)[ol[i]];
			for (int k = 0; k < c.size(); k++) {
				register uint32 v = l2a(c[k]), p = v2l(v), n = neg(p);
				if (v != cand && (cuhist[p] < p_temp || cuhist[n] < n_temp)) sp->frozen[v] = 1;
			}
		}
	}

	inline void ParaFROST::cacheCNF(const hipStream_t& s1, const hipStream_t& s2)
	{
		// 0) count clauses on GPU and variables on CPU
		countFinal();
		if (!inf.nClauses) return; // all eliminated
		if (unified_access) {
			if (profile_gpu) cutimer->start();
			// 1) compact cs w.r.t clause status on gpu
			uint32* ts = cuhist.d_lits;
			size_t tb = 0;
			DeviceSelect::If(NULL, tb, cnf->csData(), cnf->csData(), ts, cnf->size(), COMPACT_CMP(cnf)), assert(tb <= cuhist.litsbytes);
			DeviceSelect::If(ts + 1, tb, cnf->csData(), cnf->csData(), ts, cnf->size(), COMPACT_CMP(cnf), s1);
			sync();
			cnf->resize(inf.nClauses); // must be done after step (1)
			// 2) sort cs w.r.t clause size on gpu (user-enabled)
			if (sort_cnf_en) thrust::stable_sort(thrust::cuda::par.on(s1), cnf->csData(), cnf->csData() + inf.nClauses, CNF_CMP_SZ(cnf));
		}
		else {
			cumem.mirrorCNF(hcnf);
			if (profile_gpu) cutimer->start(s2);
			// 1) compact cs w.r.t  clause status on gpu
			uint32 *ts = cuhist.d_lits;
			size_t tb = 0;
			DeviceSelect::If(NULL, tb, cumem.csMem(), cumem.csMem(), ts, hcnf->size(), COMPACT_CMP(cnf)), assert(tb <= cuhist.litsbytes);
			DeviceSelect::If(ts + 1, tb, cumem.csMem(), cumem.csMem(), ts, hcnf->size(), COMPACT_CMP(cnf), s1);
			// 2) copy actual cnf data async.
			CHECK(hipMemcpyAsync(hcnf->data().mem, cumem.cnfMem(), hcnf->data().size * sizeof(S_REF), hipMemcpyDeviceToHost, s2));
			hcnf->resize(inf.nClauses); // must be done after step (2)
			// 3) sort cs w.r.t clause size on gpu (user-enabled)
			if (sort_cnf_en) thrust::stable_sort(thrust::cuda::par.on(s1), cumem.csMem(), cumem.csMem() + inf.nClauses, CNF_CMP_SZ(cnf));
			// 4) copy compact cs async.
			CHECK(hipMemcpyAsync(hcnf->csData(), cumem.csMem(), inf.nClauses * sizeof(S_REF), hipMemcpyDeviceToHost, s1));
		}
	}

	inline bool ParaFROST::enqeueCached(const hipStream_t& stream) {
		if (vars->nUnits) {
			nForced = sp->propagated;
			sync(stream); // sync units copy
			assert(vars->cachedUnits != NULL);
			uint32* t = vars->cachedUnits + vars->nUnits;
			for (uint32* u = vars->cachedUnits; u != t; u++) {
				LIT_ST val = value(*u);
				if (val == UNDEFINED) enqueue(*u);
				else if (!val) return false; // early conflict detection
			}
			if (trail.size() == sp->propagated) vars->nUnits = nForced = 0; // duplicate units
			else PFLTRAIL(this, 3);
			syncAll(); // sync ot creation
		}
		return true;
	}

	inline void	ParaFROST::cleanProped() {
		if (vars->nUnits) {
			nForced = sp->propagated - nForced;
			PFLREDALL(this, 2, "BCP Reductions");
			reduceOTAsync(cnf, ot, 0);
			nForced = 0, vars->tmpObj.clear();
			assert(vars->tmpObj.data() == cumem.unitsdPtr());
			CHECK(hipMemcpyAsync(vars->units, &vars->tmpObj, sizeof(cuVecU), hipMemcpyHostToDevice));
		}
	}

	inline void	ParaFROST::cleanDynamic() {
		if (vars != NULL) delete vars;
		cumem.freeDynamic(), vars = NULL, ot = NULL, cnf = NULL;
	}

	inline void	ParaFROST::cleanFixed() {
		cumem.freeFixed();
	}

}