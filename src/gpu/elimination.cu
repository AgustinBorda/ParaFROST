/***********************************************************************[elim.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Technische Universiteit Eindhoven (TU/e).

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "solve.h"
#include "sort.h"

using namespace pFROST;
using namespace SIGmA;

void ParaFROST::VE()
{
	if (opts.ve_en) {
		if (interrupted()) killSolver();
		PFLOG2(2, "  Eliminating variables..");
		veAsync(cnf, ot, vars, streams, cumm, cuhist, stats.sigma.calls > 1);
		postVE();
		PFLREDALL(this, 2, "BVE Reductions");
	}
}

void ParaFROST::postVE()
{
	PFLOGN2(2, "   filtering out eliminated variables..");
	int n = 0, lastIdx = -1, len = vars->numPVs;
	uint32* pvs = vars->pVars->data();
	for (int i = 0; i < len; i++) {
		const uint32 x = pvs[i];
		if (ELIMINATED(x)) {
			markEliminated(RECOVERVAR(x));
			if (IS_ADDING(x) && lastIdx < i) 
				lastIdx = i;
		}
		else pvs[n++] = x;
	}
	vars->pVars->resize(n);
	PFLENDING(2, 5, "(survived: %d, last index: %d)", n, lastIdx);
	if (!atomic_ve && lastIdx != -1) {
		PFLOGN2(2, "   resizing CNF to consider added resolvents..");
		assert(n < int(vars->numPVs));
		S_REF* rref = cuhist.d_segs;
		uint32* type = cuhist.d_hist, * rpos = type + inf.maxVar;
		uint32	lastAdded = NOVAR, lastAddedPos = NOVAR;
		S_REF	lastAddedRef = GNOREF;
		CHECK(hipMemcpyAsync(&lastAdded, type + lastIdx, sizeof(uint32), hipMemcpyDeviceToHost, streams[0]));
		CHECK(hipMemcpyAsync(&lastAddedRef, rref + lastIdx, sizeof(S_REF), hipMemcpyDeviceToHost, streams[1]));
		CHECK(hipMemcpyAsync(&lastAddedPos, rpos + lastIdx, sizeof(uint32), hipMemcpyDeviceToHost, streams[2]));
		sync(streams[0]);
		assert(lastAdded < NOVAR);
		assert(RECOVERTYPE(lastAdded) < TYPE_MASK);
		const uint32 lastAddedCls = RECOVERADDEDCLS(lastAdded);
		const uint32 lastAddedLits = RECOVERADDEDLITS(lastAdded);
		assert(lastAddedCls && lastAddedCls <= ADDEDCLS_MAX);
		assert(lastAddedLits && lastAddedLits <= ADDEDLITS_MAX);
		sync(streams[1]);
		assert(lastAddedRef < GNOREF);
		const S_REF lastAddedBuckets = lastAddedLits + hc_nbuckets * lastAddedCls;
		const S_REF data_size = lastAddedBuckets + lastAddedRef;
		sync(streams[2]);
		assert(lastAddedPos < NOVAR);
		const uint32 cs_size = lastAddedCls + lastAddedPos;
		PFLENDING(2, 5, "(new clauses: %d, data: %lld)", cs_size, data_size);
		cumm.resizeCNFAsync(cnf, data_size, cs_size);
	}
	vars->numPVs = n;
}

void ParaFROST::HSE()
{
	if (opts.hse_en || opts.ve_plus_en) {
		if (interrupted()) killSolver();
		PFLOGN2(2, "  Eliminating (self)-subsumptions..");
		hseAsync(cnf, ot, vars);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "HSE Reductions");
	}
}

void ParaFROST::BCE()
{
	if (opts.bce_en) {
		if (interrupted()) killSolver();
		if (!vars->numPVs) return;
		PFLOGN2(2, " Eliminating blocked clauses..");
		bceAsync(cnf, ot, vars, cuhist.d_vorg);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "BCE Reductions");
	}
}

void ParaFROST::ERE()
{
	if (opts.ere_en) {
		if (interrupted()) killSolver();
		if (!vars->numPVs) return;
		PFLOGN2(2, " Eliminating redundances..");
		ereCls = inf.nClauses;
		ereAsync(cnf, ot, vars);
		PFLDONE(2, 5);
		PFLREDALL(this, 2, "ERE Reductions");
	}
}